#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi_3_LLE_Kuar_ins.cu
//
// Code generation for function 'Chi_3_LLE_Kuar_ins'
//

// Include files
#include "Chi_3_LLE_Kuar_ins.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "hipfft/hipfft.h"

// Function Declarations
static __global__ void Chi_3_LLE_Kuar_ins_kernel1(const creal_T Fac_minus[512],
                                                  const creal_T F_Psi[512],
                                                  creal_T Psio[256]);

static __global__ void Chi_3_LLE_Kuar_ins_kernel2(const creal_T Fac_minus[512],
                                                  const creal_T F_Psi[512],
                                                  creal_T Psie[256],
                                                  creal_T Psio[256]);

static __global__ void Chi_3_LLE_Kuar_ins_kernel3(creal_T Psie[256]);

static __global__ void Chi_3_LLE_Kuar_ins_kernel4(const creal_T Psie[256],
                                                  const creal_T Psio[256],
                                                  creal_T x[256]);

static __global__ void Chi_3_LLE_Kuar_ins_kernel5(const creal_T Psie[256],
                                                  const creal_T Psio[256],
                                                  creal_T x[256]);

static __global__ void
Chi_3_LLE_Kuar_ins_kernel6(const creal_T Psio[256], const creal_T x[256],
                           const real_T gam3o[256], const real_T gam2o[256],
                           const creal_T Fac_plus[512], creal_T k_e[512],
                           creal_T b_x[256]);

static __global__ void Chi_3_LLE_Kuar_ins_kernel7(const creal_T Psio[256],
                                                  creal_T Psie[256]);

static __global__ void
Chi_3_LLE_Kuar_ins_kernel8(const creal_T Psie[256], const creal_T x[256],
                           const real_T gam3e[256], const real_T gam2e[256],
                           const creal_T Fac_plus[512], creal_T k_e[512]);

// Function Definitions
static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel1(
    const creal_T Fac_minus[512], const creal_T F_Psi[512], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = F_Psi[k].re;
    d1 = F_Psi[k].im;
    gam2e_re = Fac_minus[k].re;
    dc1_re = Fac_minus[k].im;
    Psio[k].re = d * gam2e_re - d1 * dc1_re;
    Psio[k].im = d * dc1_re + d1 * gam2e_re;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel2(
    const creal_T Fac_minus[512], const creal_T F_Psi[512], creal_T Psie[256],
    creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d1;
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psio[k].re;
    dc1_re = Psio[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psio[k].re = gam2e_re;
    Psio[k].im = gam2e_im;
    gam2e_im = F_Psi[k + 256].re;
    d1 = F_Psi[k + 256].im;
    gam2e_re = Fac_minus[k + 256].re;
    dc1_re = Fac_minus[k + 256].im;
    Psie[k].re = gam2e_im * gam2e_re - d1 * dc1_re;
    Psie[k].im = gam2e_im * dc1_re + d1 * gam2e_re;
  }
}

static __global__
    __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel3(creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psie[k].re;
    dc1_re = Psie[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psie[k].re = gam2e_re;
    Psie[k].im = gam2e_im;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel4(
    const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_im;
    dc1_re = Psio[k].re;
    gam2e_im = -Psio[k].im;
    d = Psie[k].re;
    d1 = Psie[k].im;
    x[k].re = dc1_re * d - gam2e_im * d1;
    x[k].im = dc1_re * d1 + gam2e_im * d;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel5(
    const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = Psio[k].re;
    d1 = Psio[k].im;
    gam2e_re = hypot(d, d1);
    dc1_re = gam2e_re * gam2e_re;
    gam2e_re = hypot(Psie[k].re, Psie[k].im);
    x[k].re = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d;
    x[k].im = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d1;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel6(
    const creal_T Psio[256], const creal_T x[256], const real_T gam3o[256],
    const real_T gam2o[256], const creal_T Fac_plus[512], creal_T k_e[512],
    creal_T b_x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_im;
    //
    d = Fac_plus[k].re;
    d1 = Fac_plus[k].im;
    dc1_re = 0.0 * d - d1;
    dc1_im = 0.0 * d1 + d;
    d = gam2o[k];
    d1 = gam3o[k];
    gam2e_im = d * b_x[k].re + d1 * x[k].re;
    d = d * b_x[k].im + d1 * x[k].im;
    k_e[k].re = dc1_re * gam2e_im - dc1_im * d;
    k_e[k].im = dc1_re * d + dc1_im * gam2e_im;
    d = Psio[k].re;
    d1 = Psio[k].im;
    b_x[k].re = d * d - d1 * d1;
    b_x[k].im = d * d1 + d1 * d;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel7(
    const creal_T Psio[256], creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = hypot(Psio[k].re, Psio[k].im);
    d1 = d * d;
    gam2e_re = Psie[k].re;
    dc1_re = Psie[k].im;
    d = hypot(gam2e_re, dc1_re);
    gam2e_re *= 2.0 * d1 + d * d;
    Psie[k].re = gam2e_re;
    dc1_re *= 2.0 * d1 + d * d;
    Psie[k].im = dc1_re;
  }
}

static __global__ __launch_bounds__(256, 1) void Chi_3_LLE_Kuar_ins_kernel8(
    const creal_T Psie[256], const creal_T x[256], const real_T gam3e[256],
    const real_T gam2e[256], const creal_T Fac_plus[512], creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_re;
    d = Fac_plus[k + 256].re;
    d1 = Fac_plus[k + 256].im;
    dc1_re = 0.0 * d - d1;
    dc1_im = 0.0 * d1 + d;
    d = gam2e[k];
    d1 = gam3e[k];
    gam2e_re = d * x[k].re + d1 * Psie[k].re;
    d = d * x[k].im + d1 * Psie[k].im;
    k_e[k + 256].re = dc1_re * gam2e_re - dc1_im * d;
    k_e[k + 256].im = dc1_re * d + dc1_im * gam2e_re;
  }
}

void Chi_3_LLE_Kuar_ins(const creal_T F_Psi[512], const creal_T Fac_plus[512],
                        const creal_T Fac_minus[512], const real_T gam2o[256],
                        const real_T gam2e[256], const real_T gam3o[256],
                        const real_T gam3e[256], const real_T[512],
                        creal_T k_e[512])
{
  hipfftHandle b_fftPlanHandle;
  hipfftHandle c_fftPlanHandle;
  hipfftHandle d_fftPlanHandle;
  hipfftHandle e_fftPlanHandle;
  hipfftHandle f_fftPlanHandle;
  hipfftHandle fftPlanHandle;
  creal_T(*b_gpu_x)[256];
  creal_T(*gpu_Psie)[256];
  creal_T(*gpu_Psio)[256];
  creal_T(*gpu_x)[256];
  int32_T inembed;
  hipMalloc(&b_gpu_x, 4096UL);
  hipMalloc(&gpu_x, 4096UL);
  hipMalloc(&gpu_Psie, 4096UL);
  hipMalloc(&gpu_Psio, 4096UL);
  Chi_3_LLE_Kuar_ins_kernel1<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      Fac_minus, F_Psi, *gpu_Psio);
  inembed = 256;
  fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psio)[0],
               (hipfftDoubleComplex *)&(*gpu_Psio)[0], HIPFFT_BACKWARD);
  Chi_3_LLE_Kuar_ins_kernel2<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      Fac_minus, F_Psi, *gpu_Psie, *gpu_Psio);
  inembed = 256;
  b_fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
               (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_BACKWARD);
  Chi_3_LLE_Kuar_ins_kernel3<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psie);
  Chi_3_LLE_Kuar_ins_kernel4<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psie, *gpu_Psio, *gpu_x);
  inembed = 256;
  c_fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(c_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
               (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD);
  Chi_3_LLE_Kuar_ins_kernel5<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psie, *gpu_Psio, *b_gpu_x);
  inembed = 256;
  d_fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(d_fftPlanHandle, (hipfftDoubleComplex *)&(*b_gpu_x)[0],
               (hipfftDoubleComplex *)&(*b_gpu_x)[0], HIPFFT_FORWARD);
  //
  Chi_3_LLE_Kuar_ins_kernel6<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psio, *b_gpu_x, gam3o, gam2o, Fac_plus, k_e, *gpu_x);
  inembed = 256;
  e_fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(e_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
               (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD);
  Chi_3_LLE_Kuar_ins_kernel7<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psio, *gpu_Psie);
  inembed = 256;
  f_fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
  hipfftExecZ2Z(f_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
               (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_FORWARD);
  Chi_3_LLE_Kuar_ins_kernel8<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
      *gpu_Psie, *gpu_x, gam3e, gam2e, Fac_plus, k_e);
  //
  //     k_e(1)      = k_e(1) + ;
  hipFree(*gpu_Psio);
  hipFree(*gpu_Psie);
  hipFree(*gpu_x);
  hipFree(*b_gpu_x);
}

// End of code generation (Chi_3_LLE_Kuar_ins.cu)

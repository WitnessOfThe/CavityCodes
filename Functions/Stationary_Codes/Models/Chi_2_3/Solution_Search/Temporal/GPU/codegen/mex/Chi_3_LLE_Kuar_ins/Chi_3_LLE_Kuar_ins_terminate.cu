//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi_3_LLE_Kuar_ins_terminate.cu
//
// Code generation for function 'Chi_3_LLE_Kuar_ins_terminate'
//

// Include files
#include "Chi_3_LLE_Kuar_ins_terminate.h"
#include "Chi_3_LLE_Kuar_ins_data.h"
#include "_coder_Chi_3_LLE_Kuar_ins_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Chi_3_LLE_Kuar_ins_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void Chi_3_LLE_Kuar_ins_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (Chi_3_LLE_Kuar_ins_terminate.cu)

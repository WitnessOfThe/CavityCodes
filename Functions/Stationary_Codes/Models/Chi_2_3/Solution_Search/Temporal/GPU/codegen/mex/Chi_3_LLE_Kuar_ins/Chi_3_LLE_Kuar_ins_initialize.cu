//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi_3_LLE_Kuar_ins_initialize.cu
//
// Code generation for function 'Chi_3_LLE_Kuar_ins_initialize'
//

// Include files
#include "Chi_3_LLE_Kuar_ins_initialize.h"
#include "Chi_3_LLE_Kuar_ins_data.h"
#include "_coder_Chi_3_LLE_Kuar_ins_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Chi_3_LLE_Kuar_ins_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal,
                          (const char_T *)"distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

// End of code generation (Chi_3_LLE_Kuar_ins_initialize.cu)

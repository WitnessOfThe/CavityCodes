//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi23_Runge_IntegrationSteps_terminate.cu
//
// Code generation for function 'Chi23_Runge_IntegrationSteps_terminate'
//

// Include files
#include "Chi23_Runge_IntegrationSteps_terminate.h"
#include "Chi23_Runge_IntegrationSteps_data.h"
#include "_coder_Chi23_Runge_IntegrationSteps_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Chi23_Runge_IntegrationSteps_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void Chi23_Runge_IntegrationSteps_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (Chi23_Runge_IntegrationSteps_terminate.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi23_Runge_IntegrationSteps.cu
//
// Code generation for function 'Chi23_Runge_IntegrationSteps'
//

// Include files
#include "Chi23_Runge_IntegrationSteps.h"
#include "Chi23_Runge_IntegrationSteps_data.h"
#include "Chi23_Runge_IntegrationSteps_types.h"
#include "rt_nonfinite.h"
#include "MWCudaDimUtility.hpp"
#include <cstdio>
#include <cstdlib>
#include <cstring>

// Variable Definitions
static emlrtMCInfo emlrtMCI{
    21,                   // lineNo
    16,                   // colNo
    "Runge_Kuarong_step", // fName
    "/home/dp710/NewGitFolder/CavityCodes/Functions/Stationary_Codes/Models/"
    "Chi_2_3/Solution_Search/Temporal/GPU/Chi23_Runge_Integrat"
    "ionSteps.m" // pName
};

static emlrtMCInfo b_emlrtMCI{
    35,                   // lineNo
    21,                   // colNo
    "Runge_Kuarong_step", // fName
    "/home/dp710/NewGitFolder/CavityCodes/Functions/Stationary_Codes/Models/"
    "Chi_2_3/Solution_Search/Temporal/GPU/Chi23_Runge_Integrat"
    "ionSteps.m" // pName
};

// Function Declarations
static const mxArray *
Chi_3_LLE_Kuar_ins_mex(const mxArray *m1, const mxArray *m2, const mxArray *m3,
                       const mxArray *m4, const mxArray *m5, const mxArray *m6,
                       const mxArray *m7, const mxArray *m8,
                       emlrtMCInfo *location);

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[512]);

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line);

static __global__ void d_Chi23_Runge_IntegrationSteps_(const creal_T d[1536],
                                                       creal_T b_d[1536]);

static __global__ void e_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                       creal_T E_temp[512]);

static void emlrt_marshallIn(const mxArray *c_a__output_of_Chi_3_LLE_Kuar_i,
                             const char_T *identifier, creal_T y[512]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             creal_T y[512]);

static __global__ void
f_Chi23_Runge_IntegrationSteps_(const creal_T exp_plus_omega[1536],
                                creal_T u[512]);

static __global__ void
g_Chi23_Runge_IntegrationSteps_(const creal_T exp_minus_omega[1536],
                                creal_T u[512]);

static __global__ void h_Chi23_Runge_IntegrationSteps_(const creal_T u[512],
                                                       creal_T d[1536]);

static __global__ void
i_Chi23_Runge_IntegrationSteps_(const creal_T d[1536], const int32_T ii,
                                const real_T dt, const creal_T E_temp[512],
                                const real_T mode_range[512], creal_T u[512]);

static __global__ void j_Chi23_Runge_IntegrationSteps_(
    const creal_T u[512], const real_T mode_range[512], creal_T E_temp[512]);

static __global__ void
k_Chi23_Runge_IntegrationSteps_(const creal_T exp_plus_omega[1536],
                                const int32_T i, creal_T u[512]);

static __global__ void
l_Chi23_Runge_IntegrationSteps_(const creal_T exp_minus_omega[1536],
                                const int32_T i, creal_T u[512]);

static __global__ void m_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                       const creal_T u[512],
                                                       const int32_T i,
                                                       creal_T E_temp[512],
                                                       creal_T d[1536]);

static __global__ void
n_Chi23_Runge_IntegrationSteps_(const creal_T d[1536], const int32_T i6,
                                const real_T dt, const creal_T F_e[512],
                                const real_T mode_range[512], creal_T u[512]);

static __global__ void
o_Chi23_Runge_IntegrationSteps_(const creal_T u[512],
                                const real_T mode_range[512], creal_T F_e[512]);

static __global__ void
p_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                const creal_T shift_back[512],
                                const real_T mode_range[512], creal_T u[512]);

static __global__ void
q_Chi23_Runge_IntegrationSteps_(const creal_T u[512],
                                const real_T mode_range[512], creal_T F_e[512]);

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString);

// Function Definitions
static const mxArray *
Chi_3_LLE_Kuar_ins_mex(const mxArray *m1, const mxArray *m2, const mxArray *m3,
                       const mxArray *m4, const mxArray *m5, const mxArray *m6,
                       const mxArray *m7, const mxArray *m8,
                       emlrtMCInfo *location)
{
  const mxArray *pArrays[8];
  const mxArray *m;
  pArrays[0] = m1;
  pArrays[1] = m2;
  pArrays[2] = m3;
  pArrays[3] = m4;
  pArrays[4] = m5;
  pArrays[5] = m6;
  pArrays[6] = m7;
  pArrays[7] = m8;
  return emlrtCallMATLABR2012b(emlrtRootTLSGlobal, 1, &m, 8, &pArrays[0],
                               (const char_T *)"Chi_3_LLE_Kuar_ins_mex", true,
                               location);
}

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[512])
{
  static const int32_T dims[2]{1, 512};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", true, 2U, (void *)&dims[0]);
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 8, true);
  emlrtDestroyArray(&src);
}

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
}

static __global__ __launch_bounds__(
    512, 1) void d_Chi23_Runge_IntegrationSteps_(const creal_T d[1536],
                                                 creal_T b_d[1536])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 1536) {
    b_d[i] = d[i];
  }
}

static __global__ __launch_bounds__(
    512, 1) void e_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                 creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    E_temp[i] = F_e[i];
  }
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, creal_T y[512])
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *c_a__output_of_Chi_3_LLE_Kuar_i,
                             const char_T *identifier, creal_T y[512])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(c_a__output_of_Chi_3_LLE_Kuar_i), &thisId, y);
  emlrtDestroyArray(&c_a__output_of_Chi_3_LLE_Kuar_i);
}

static __global__
    __launch_bounds__(512, 1) void f_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_plus_omega[1536], creal_T u[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    u[i] = exp_plus_omega[3 * i];
  }
}

static __global__
    __launch_bounds__(512, 1) void g_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_minus_omega[1536], creal_T u[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    u[i] = exp_minus_omega[3 * i];
  }
}

static __global__ __launch_bounds__(
    512, 1) void h_Chi23_Runge_IntegrationSteps_(const creal_T u[512],
                                                 creal_T d[1536])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    d[3 * i] = u[i];
  }
}

static __global__
    __launch_bounds__(512, 1) void i_Chi23_Runge_IntegrationSteps_(
        const creal_T d[1536], const int32_T ii, const real_T dt,
        const creal_T E_temp[512], const real_T mode_range[512], creal_T u[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    real_T b_d;
    b_d = mode_range[i];
    u[i].re = E_temp[static_cast<int32_T>(b_d) - 1].re +
              dt * d[ii + 3 * (static_cast<int32_T>(b_d) - 1)].re;
    u[i].im = E_temp[static_cast<int32_T>(b_d) - 1].im +
              dt * d[ii + 3 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__
    __launch_bounds__(512, 1) void j_Chi23_Runge_IntegrationSteps_(
        const creal_T u[512], const real_T mode_range[512], creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    E_temp[static_cast<int32_T>(mode_range[i]) - 1] = u[i];
  }
}

static __global__
    __launch_bounds__(512, 1) void k_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_plus_omega[1536], const int32_T i, creal_T u[512])
{
  uint64_T threadId;
  int32_T b_i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 512) {
    u[b_i] = exp_plus_omega[i + 3 * b_i];
  }
}

static __global__
    __launch_bounds__(512, 1) void l_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_minus_omega[1536], const int32_T i, creal_T u[512])
{
  uint64_T threadId;
  int32_T b_i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 512) {
    u[b_i] = exp_minus_omega[i + 3 * b_i];
  }
}

static __global__
    __launch_bounds__(512, 1) void m_Chi23_Runge_IntegrationSteps_(
        const creal_T F_e[512], const creal_T u[512], const int32_T i,
        creal_T E_temp[512], creal_T d[1536])
{
  uint64_T threadId;
  int32_T b_i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  b_i = static_cast<int32_T>(threadId);
  if (b_i < 512) {
    d[i + 3 * b_i] = u[b_i];
    E_temp[b_i] = F_e[b_i];
  }
}

static __global__
    __launch_bounds__(512, 1) void n_Chi23_Runge_IntegrationSteps_(
        const creal_T d[1536], const int32_T i6, const real_T dt,
        const creal_T F_e[512], const real_T mode_range[512], creal_T u[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    real_T b_d;
    b_d = mode_range[i];
    u[i].re = F_e[static_cast<int32_T>(b_d) - 1].re +
              dt * d[i6 + 3 * (static_cast<int32_T>(b_d) - 1)].re;
    u[i].im = F_e[static_cast<int32_T>(b_d) - 1].im +
              dt * d[i6 + 3 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__
    __launch_bounds__(512, 1) void o_Chi23_Runge_IntegrationSteps_(
        const creal_T u[512], const real_T mode_range[512], creal_T F_e[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    F_e[static_cast<int32_T>(mode_range[i]) - 1] = u[i];
  }
}

static __global__ __launch_bounds__(
    512, 1) void p_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                 const creal_T shift_back[512],
                                                 const real_T mode_range[512],
                                                 creal_T u[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    real_T d;
    d = mode_range[i];
    u[i].re = shift_back[static_cast<int32_T>(d) - 1].re *
                  F_e[static_cast<int32_T>(d) - 1].re -
              shift_back[static_cast<int32_T>(d) - 1].im *
                  F_e[static_cast<int32_T>(d) - 1].im;
    u[i].im = shift_back[static_cast<int32_T>(d) - 1].re *
                  F_e[static_cast<int32_T>(d) - 1].im +
              shift_back[static_cast<int32_T>(d) - 1].im *
                  F_e[static_cast<int32_T>(d) - 1].re;
  }
}

static __global__
    __launch_bounds__(512, 1) void q_Chi23_Runge_IntegrationSteps_(
        const creal_T u[512], const real_T mode_range[512], creal_T F_e[512])
{
  uint64_T threadId;
  int32_T i;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  i = static_cast<int32_T>(threadId);
  if (i < 512) {
    F_e[static_cast<int32_T>(mode_range[i]) - 1] = u[i];
  }
}

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint32_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = (char_T *)calloc(len + 1U, 1U);
  fn = (char_T *)calloc(len + 1U, 1U);
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(errCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
}

void Chi23_Runge_IntegrationSteps(
    real_T nt, real_T dt, const creal_T d[1536], creal_T F_e[512],
    const creal_T exp_plus_omega[1536], const creal_T exp_minus_omega[1536],
    const struct0_T *Runge, const creal_T shift_back[512],
    const real_T gam2o[256], const real_T gam2e[256], const real_T gam3o[256],
    const real_T gam3e[256], const real_T H_f[512],
    const real_T mode_range[512])
{
  static const int32_T iv[2]{1, 512};
  static const int32_T iv1[2]{1, 512};
  static const int32_T iv10[2]{1, 512};
  static const int32_T iv11[2]{1, 256};
  static const int32_T iv12[2]{1, 256};
  static const int32_T iv13[2]{1, 256};
  static const int32_T iv14[2]{1, 256};
  static const int32_T iv15[2]{1, 512};
  static const int32_T iv2[2]{1, 512};
  static const int32_T iv3[2]{1, 256};
  static const int32_T iv4[2]{1, 256};
  static const int32_T iv5[2]{1, 256};
  static const int32_T iv6[2]{1, 256};
  static const int32_T iv7[2]{1, 512};
  static const int32_T iv8[2]{1, 512};
  static const int32_T iv9[2]{1, 512};
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *e_y;
  const mxArray *f_y;
  const mxArray *g_y;
  const mxArray *h_y;
  const mxArray *i_y;
  const mxArray *j_y;
  const mxArray *k_y;
  const mxArray *l_y;
  const mxArray *m;
  const mxArray *m_y;
  const mxArray *n_y;
  const mxArray *o_y;
  const mxArray *p_y;
  const mxArray *y;
  creal_T(*gpu_d)[1536];
  creal_T E_temp[512];
  creal_T u[512];
  creal_T(*cpu_F_e)[512];
  creal_T(*gpu_E_temp)[512];
  creal_T(*gpu_u)[512];
  real_T cpu_H_f[512];
  real_T(*gpu_mode_range)[512];
  real_T cpu_gam2e[256];
  real_T cpu_gam2o[256];
  real_T cpu_gam3e[256];
  real_T cpu_gam3o[256];
  real_T *pData;
  int32_T i;
  boolean_T cpu_H_f_dirtyOnGpu;
  boolean_T cpu_gam2e_dirtyOnGpu;
  boolean_T cpu_gam2o_dirtyOnGpu;
  boolean_T cpu_gam3e_dirtyOnGpu;
  boolean_T cpu_gam3o_dirtyOnGpu;
  boolean_T mode_range_dirtyOnCpu;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  hipMalloc(&gpu_mode_range, 4096UL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_u, 8192UL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_E_temp, 8192UL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_d, 24576UL);
  CUDACHECK(hipGetLastError());
  cpu_F_e = (creal_T(*)[512])malloc(8192U);
  cpu_H_f_dirtyOnGpu = true;
  cpu_gam3e_dirtyOnGpu = true;
  cpu_gam3o_dirtyOnGpu = true;
  cpu_gam2e_dirtyOnGpu = true;
  cpu_gam2o_dirtyOnGpu = true;
  mode_range_dirtyOnCpu = true;
  i = static_cast<int32_T>(nt);
  for (int32_T ni{0}; ni < i; ni++) {
    real_T b_dt;
    int32_T b_i;
    int32_T i6;
    d_Chi23_Runge_IntegrationSteps_<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        d, *gpu_d);
    CUDACHECK(hipGetLastError());
    e_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        F_e, *gpu_E_temp);
    CUDACHECK(hipGetLastError());
    y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv[0], mxDOUBLE_CLASS,
                                mxCOMPLEX);
    hipMemcpy(*cpu_F_e, F_e, 8192UL, hipMemcpyDeviceToHost);
    CUDACHECK(hipGetLastError());
    emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &(*cpu_F_e)[0], 8);
    emlrtAssign(&y, m);
    f_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        exp_plus_omega, *gpu_u);
    CUDACHECK(hipGetLastError());
    b_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv1[0], mxDOUBLE_CLASS,
                                mxCOMPLEX);
    hipMemcpy(u, *gpu_u, 8192UL, hipMemcpyDeviceToHost);
    CUDACHECK(hipGetLastError());
    emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u[0], 8);
    emlrtAssign(&b_y, m);
    g_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        exp_minus_omega, *gpu_u);
    CUDACHECK(hipGetLastError());
    c_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv2[0], mxDOUBLE_CLASS,
                                mxCOMPLEX);
    hipMemcpy(u, *gpu_u, 8192UL, hipMemcpyDeviceToHost);
    CUDACHECK(hipGetLastError());
    emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u[0], 8);
    emlrtAssign(&c_y, m);
    d_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv3[0], mxDOUBLE_CLASS,
                                mxREAL);
    pData = emlrtMxGetPr(m);
    for (b_i = 0; b_i < 256; b_i++) {
      if (cpu_gam2o_dirtyOnGpu) {
        hipMemcpy(cpu_gam2o, gam2o, 2048UL, hipMemcpyDeviceToHost);
        CUDACHECK(hipGetLastError());
      }
      cpu_gam2o_dirtyOnGpu = false;
      pData[b_i] = cpu_gam2o[b_i];
    }
    emlrtAssign(&d_y, m);
    e_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv4[0], mxDOUBLE_CLASS,
                                mxREAL);
    pData = emlrtMxGetPr(m);
    for (b_i = 0; b_i < 256; b_i++) {
      if (cpu_gam2e_dirtyOnGpu) {
        hipMemcpy(cpu_gam2e, gam2e, 2048UL, hipMemcpyDeviceToHost);
        CUDACHECK(hipGetLastError());
      }
      cpu_gam2e_dirtyOnGpu = false;
      pData[b_i] = cpu_gam2e[b_i];
    }
    emlrtAssign(&e_y, m);
    f_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv5[0], mxDOUBLE_CLASS,
                                mxREAL);
    pData = emlrtMxGetPr(m);
    for (b_i = 0; b_i < 256; b_i++) {
      if (cpu_gam3o_dirtyOnGpu) {
        hipMemcpy(cpu_gam3o, gam3o, 2048UL, hipMemcpyDeviceToHost);
        CUDACHECK(hipGetLastError());
      }
      cpu_gam3o_dirtyOnGpu = false;
      pData[b_i] = cpu_gam3o[b_i];
    }
    emlrtAssign(&f_y, m);
    g_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv6[0], mxDOUBLE_CLASS,
                                mxREAL);
    pData = emlrtMxGetPr(m);
    for (b_i = 0; b_i < 256; b_i++) {
      if (cpu_gam3e_dirtyOnGpu) {
        hipMemcpy(cpu_gam3e, gam3e, 2048UL, hipMemcpyDeviceToHost);
        CUDACHECK(hipGetLastError());
      }
      cpu_gam3e_dirtyOnGpu = false;
      pData[b_i] = cpu_gam3e[b_i];
    }
    emlrtAssign(&g_y, m);
    h_y = nullptr;
    m = emlrtCreateNumericArray(2, (const void *)&iv7[0], mxDOUBLE_CLASS,
                                mxREAL);
    pData = emlrtMxGetPr(m);
    for (b_i = 0; b_i < 512; b_i++) {
      if (cpu_H_f_dirtyOnGpu) {
        hipMemcpy(cpu_H_f, H_f, 4096UL, hipMemcpyDeviceToHost);
        CUDACHECK(hipGetLastError());
      }
      cpu_H_f_dirtyOnGpu = false;
      pData[b_i] = cpu_H_f[b_i];
    }
    emlrtAssign(&h_y, m);
    emlrt_marshallIn(emlrtCoerceToClassR2014b(
                         Chi_3_LLE_Kuar_ins_mex(y, b_y, c_y, d_y, e_y, f_y, g_y,
                                                h_y, &emlrtMCI),
                         (char_T *)"double"),
                     "<output of Chi_3_LLE_Kuar_ins_mex>", u);
    hipMemcpy(*gpu_u, u, 8192UL, hipMemcpyHostToDevice);
    CUDACHECK(hipGetLastError());
    h_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_u, *gpu_d);
    CUDACHECK(hipGetLastError());
    i6 = static_cast<int32_T>(Runge->s + -1.0);
    for (int32_T i2{0}; i2 < i6; i2++) {
      b_i = static_cast<int32_T>(Runge->s - 1.0);
      for (int32_T ii{0}; ii < b_i; ii++) {
        b_dt = Runge->b[(i2 + 3 * ii) + 1];
        if (b_dt != 0.0) {
          b_dt *= dt;
          if (mode_range_dirtyOnCpu) {
            hipMemcpy(*gpu_mode_range, mode_range, 4096UL,
                       hipMemcpyHostToDevice);
            CUDACHECK(hipGetLastError());
          }
          i_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              *gpu_d, ii, b_dt, *gpu_E_temp, *gpu_mode_range, *gpu_u);
          CUDACHECK(hipGetLastError());
          mode_range_dirtyOnCpu = false;
          j_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              *gpu_u, *gpu_mode_range, *gpu_E_temp);
          CUDACHECK(hipGetLastError());
        }
      }
      i_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv8[0], mxDOUBLE_CLASS,
                                  mxCOMPLEX);
      hipMemcpy(E_temp, *gpu_E_temp, 8192UL, hipMemcpyDeviceToHost);
      CUDACHECK(hipGetLastError());
      emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &E_temp[0], 8);
      emlrtAssign(&i_y, m);
      k_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          exp_plus_omega, i2 + 1, *gpu_u);
      CUDACHECK(hipGetLastError());
      j_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv9[0], mxDOUBLE_CLASS,
                                  mxCOMPLEX);
      hipMemcpy(u, *gpu_u, 8192UL, hipMemcpyDeviceToHost);
      CUDACHECK(hipGetLastError());
      emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u[0], 8);
      emlrtAssign(&j_y, m);
      l_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          exp_minus_omega, i2 + 1, *gpu_u);
      CUDACHECK(hipGetLastError());
      k_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv10[0], mxDOUBLE_CLASS,
                                  mxCOMPLEX);
      hipMemcpy(u, *gpu_u, 8192UL, hipMemcpyDeviceToHost);
      CUDACHECK(hipGetLastError());
      emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u[0], 8);
      emlrtAssign(&k_y, m);
      l_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv11[0], mxDOUBLE_CLASS,
                                  mxREAL);
      pData = emlrtMxGetPr(m);
      for (b_i = 0; b_i < 256; b_i++) {
        if (cpu_gam2o_dirtyOnGpu) {
          hipMemcpy(cpu_gam2o, gam2o, 2048UL, hipMemcpyDeviceToHost);
          CUDACHECK(hipGetLastError());
        }
        cpu_gam2o_dirtyOnGpu = false;
        pData[b_i] = cpu_gam2o[b_i];
      }
      emlrtAssign(&l_y, m);
      m_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv12[0], mxDOUBLE_CLASS,
                                  mxREAL);
      pData = emlrtMxGetPr(m);
      for (b_i = 0; b_i < 256; b_i++) {
        if (cpu_gam2e_dirtyOnGpu) {
          hipMemcpy(cpu_gam2e, gam2e, 2048UL, hipMemcpyDeviceToHost);
          CUDACHECK(hipGetLastError());
        }
        cpu_gam2e_dirtyOnGpu = false;
        pData[b_i] = cpu_gam2e[b_i];
      }
      emlrtAssign(&m_y, m);
      n_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv13[0], mxDOUBLE_CLASS,
                                  mxREAL);
      pData = emlrtMxGetPr(m);
      for (b_i = 0; b_i < 256; b_i++) {
        if (cpu_gam3o_dirtyOnGpu) {
          hipMemcpy(cpu_gam3o, gam3o, 2048UL, hipMemcpyDeviceToHost);
          CUDACHECK(hipGetLastError());
        }
        cpu_gam3o_dirtyOnGpu = false;
        pData[b_i] = cpu_gam3o[b_i];
      }
      emlrtAssign(&n_y, m);
      o_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv14[0], mxDOUBLE_CLASS,
                                  mxREAL);
      pData = emlrtMxGetPr(m);
      for (b_i = 0; b_i < 256; b_i++) {
        if (cpu_gam3e_dirtyOnGpu) {
          hipMemcpy(cpu_gam3e, gam3e, 2048UL, hipMemcpyDeviceToHost);
          CUDACHECK(hipGetLastError());
        }
        cpu_gam3e_dirtyOnGpu = false;
        pData[b_i] = cpu_gam3e[b_i];
      }
      emlrtAssign(&o_y, m);
      p_y = nullptr;
      m = emlrtCreateNumericArray(2, (const void *)&iv15[0], mxDOUBLE_CLASS,
                                  mxREAL);
      pData = emlrtMxGetPr(m);
      for (b_i = 0; b_i < 512; b_i++) {
        if (cpu_H_f_dirtyOnGpu) {
          hipMemcpy(cpu_H_f, H_f, 4096UL, hipMemcpyDeviceToHost);
          CUDACHECK(hipGetLastError());
        }
        cpu_H_f_dirtyOnGpu = false;
        pData[b_i] = cpu_H_f[b_i];
      }
      emlrtAssign(&p_y, m);
      emlrt_marshallIn(emlrtCoerceToClassR2014b(
                           Chi_3_LLE_Kuar_ins_mex(i_y, j_y, k_y, l_y, m_y, n_y,
                                                  o_y, p_y, &b_emlrtMCI),
                           (char_T *)"double"),
                       "<output of Chi_3_LLE_Kuar_ins_mex>", u);
      hipMemcpy(*gpu_u, u, 8192UL, hipMemcpyHostToDevice);
      CUDACHECK(hipGetLastError());
      m_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          F_e, *gpu_u, i2 + 1, *gpu_E_temp, *gpu_d);
      CUDACHECK(hipGetLastError());
    }
    b_i = static_cast<int32_T>(Runge->s);
    for (i6 = 0; i6 < b_i; i6++) {
      b_dt = Runge->c[i6];
      if (b_dt != 0.0) {
        b_dt *= dt;
        if (mode_range_dirtyOnCpu) {
          hipMemcpy(*gpu_mode_range, mode_range, 4096UL,
                     hipMemcpyHostToDevice);
          CUDACHECK(hipGetLastError());
        }
        n_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            *gpu_d, i6, b_dt, F_e, *gpu_mode_range, *gpu_u);
        CUDACHECK(hipGetLastError());
        mode_range_dirtyOnCpu = false;
        o_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            *gpu_u, *gpu_mode_range, F_e);
        CUDACHECK(hipGetLastError());
      }
    }
    if (mode_range_dirtyOnCpu) {
      hipMemcpy(*gpu_mode_range, mode_range, 4096UL, hipMemcpyHostToDevice);
      CUDACHECK(hipGetLastError());
    }
    p_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        F_e, shift_back, *gpu_mode_range, *gpu_u);
    CUDACHECK(hipGetLastError());
    mode_range_dirtyOnCpu = false;
    q_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_u, *gpu_mode_range, F_e);
    CUDACHECK(hipGetLastError());
    // ,Runge,exp_plus_omega,exp_minus_omega,Temp,shift_back
  }
  free(*cpu_F_e);
  hipFree(*gpu_d);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_E_temp);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_u);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_mode_range);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

// End of code generation (Chi23_Runge_IntegrationSteps.cu)

#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi23_Runge_IntegrationSteps.cu
//
// Code generation for function 'Chi23_Runge_IntegrationSteps'
//

// Include files
#include "Chi23_Runge_IntegrationSteps.h"
#include "Chi23_Runge_IntegrationSteps_data.h"
#include "Chi23_Runge_IntegrationSteps_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWErrorCodeUtils.hpp"
#include "hipfft/hipfft.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>

// Function Declarations
static __global__ void ab_Chi23_Runge_IntegrationSteps(
    const creal_T Psio[256], const creal_T x[256], const real_T gam3o[256],
    const real_T gam2o[256], const creal_T exp_plus_omega[1536],
    const int32_T inembed, creal_T k_e[512], creal_T b_x[256]);

static void b_raiseCudaError(uint32_T errCode, const char_T *file,
                             uint32_T line, const char_T *errorName,
                             const char_T *errorString);

static __global__ void bb_Chi23_Runge_IntegrationSteps(const creal_T Psio[256],
                                                       creal_T Psie[256]);

static __global__ void cb_Chi23_Runge_IntegrationSteps(
    const creal_T Psie[256], const creal_T x[256], const real_T gam3e[256],
    const real_T gam2e[256], const creal_T exp_plus_omega[1536],
    const int32_T inembed, creal_T k_e[512]);

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line);

static void cufftCheck(hipfftResult_t errCode, const char_T *file,
                       uint32_T line);

static __global__ void
d_Chi23_Runge_IntegrationSteps_(const creal_T exp_plus_omega[1536],
                                creal_T *b_exp_plus_omega);

static __global__ void db_Chi23_Runge_IntegrationSteps(const creal_T F_e[512],
                                                       const creal_T k_e[512],
                                                       const int32_T inembed,
                                                       creal_T E_temp[512],
                                                       creal_T d[1536]);

static __global__ void e_Chi23_Runge_IntegrationSteps_(const creal_T d[1536],
                                                       creal_T b_d[1536]);

static __global__ void eb_Chi23_Runge_IntegrationSteps(
    const creal_T d[1536], const int32_T inembed, const real_T dt,
    const creal_T F_e[512], const real_T mode_range[512], creal_T E_temp[512]);

static __global__ void f_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                       creal_T E_temp[512]);

static __global__ void
fb_Chi23_Runge_IntegrationSteps(const creal_T E_temp[512],
                                const real_T mode_range[512], creal_T F_e[512]);

static __global__ void
g_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                const creal_T exp_minus_omega[1536],
                                creal_T Psio[256]);

static __global__ void gb_Chi23_Runge_IntegrationSteps(
    const creal_T F_e[512], const creal_T shift_back[512],
    const real_T mode_range[512], creal_T E_temp[512]);

static __global__ void
h_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                const creal_T exp_minus_omega[1536],
                                creal_T Psie[256], creal_T Psio[256]);

static __global__ void
hb_Chi23_Runge_IntegrationSteps(const creal_T E_temp[512],
                                const real_T mode_range[512], creal_T F_e[512]);

static __global__ void i_Chi23_Runge_IntegrationSteps_(creal_T Psie[256]);

static __global__ void
j_Chi23_Runge_IntegrationSteps_(const creal_T *exp_plus_omega,
                                const real_T H_f[512], creal_T k_e[512]);

static __global__ void k_Chi23_Runge_IntegrationSteps_(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void l_Chi23_Runge_IntegrationSteps_(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void m_Chi23_Runge_IntegrationSteps_(
    const creal_T Psio[256], const creal_T x[256], const real_T gam3o[256],
    const real_T gam2o[256], const creal_T exp_plus_omega[1536],
    creal_T k_e[512], creal_T b_x[256]);

static __global__ void n_Chi23_Runge_IntegrationSteps_(const creal_T Psio[256],
                                                       creal_T Psie[256]);

static __global__ void o_Chi23_Runge_IntegrationSteps_(
    const creal_T Psie[256], const creal_T x[256], const real_T gam3e[256],
    const real_T gam2e[256], const creal_T exp_plus_omega[1536],
    creal_T k_e[512]);

static __global__ void p_Chi23_Runge_IntegrationSteps_(const creal_T k_e[512],
                                                       creal_T d[1536]);

static __global__ void
q_Chi23_Runge_IntegrationSteps_(const creal_T d[1536], const int32_T inembed,
                                const real_T dt, const creal_T E_temp[512],
                                const real_T mode_range[512],
                                creal_T b_E_temp[512]);

static __global__ void
r_Chi23_Runge_IntegrationSteps_(const creal_T E_temp[512],
                                const real_T mode_range[512],
                                creal_T b_E_temp[512]);

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString);

static __global__ void
s_Chi23_Runge_IntegrationSteps_(const creal_T exp_minus_omega[1536],
                                const int32_T inembed,
                                const creal_T E_temp[512], creal_T Psio[256]);

static __global__ void t_Chi23_Runge_IntegrationSteps_(
    const creal_T exp_minus_omega[1536], const int32_T inembed,
    const creal_T E_temp[512], creal_T Psie[256], creal_T Psio[256]);

static __global__ void u_Chi23_Runge_IntegrationSteps_(creal_T Psie[256]);

static __global__ void
v_Chi23_Runge_IntegrationSteps_(const creal_T exp_plus_omega[1536],
                                const int32_T i2, creal_T *b_exp_plus_omega);

static __global__ void
w_Chi23_Runge_IntegrationSteps_(const creal_T *exp_plus_omega,
                                const real_T H_f[512], creal_T k_e[512]);

static __global__ void x_Chi23_Runge_IntegrationSteps_(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void y_Chi23_Runge_IntegrationSteps_(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

// Function Definitions
static __global__
    __launch_bounds__(256, 1) void ab_Chi23_Runge_IntegrationSteps(
        const creal_T Psio[256], const creal_T x[256], const real_T gam3o[256],
        const real_T gam2o[256], const creal_T exp_plus_omega[1536],
        const int32_T inembed, creal_T k_e[512], creal_T b_x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_im;
    //
    dc1_re = 0.0 * exp_plus_omega[inembed + 3 * k].re -
             exp_plus_omega[inembed + 3 * k].im;
    dc1_im = 0.0 * exp_plus_omega[inembed + 3 * k].im +
             exp_plus_omega[inembed + 3 * k].re;
    d = gam2o[k];
    d1 = gam3o[k];
    gam2e_im = d * b_x[k].re + d1 * x[k].re;
    d = d * b_x[k].im + d1 * x[k].im;
    k_e[k].re = dc1_re * gam2e_im - dc1_im * d;
    k_e[k].im = dc1_re * d + dc1_im * gam2e_im;
    d = Psio[k].re;
    d1 = Psio[k].im;
    b_x[k].re = d * d - d1 * d1;
    b_x[k].im = d * d1 + d1 * d;
  }
}

static void b_raiseCudaError(uint32_T errCode, const char_T *file,
                             uint32_T line, const char_T *errorName,
                             const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint32_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = (char_T *)calloc(len + 1U, 1U);
  fn = (char_T *)calloc(len + 1U, 1U);
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(errCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
}

static __global__ __launch_bounds__(
    256, 1) void bb_Chi23_Runge_IntegrationSteps(const creal_T Psio[256],
                                                 creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = hypot(Psio[k].re, Psio[k].im);
    d1 = d * d;
    gam2e_re = Psie[k].re;
    dc1_re = Psie[k].im;
    d = hypot(gam2e_re, dc1_re);
    gam2e_re *= 2.0 * d1 + d * d;
    Psie[k].re = gam2e_re;
    dc1_re *= 2.0 * d1 + d * d;
    Psie[k].im = dc1_re;
  }
}

static __global__
    __launch_bounds__(256, 1) void cb_Chi23_Runge_IntegrationSteps(
        const creal_T Psie[256], const creal_T x[256], const real_T gam3e[256],
        const real_T gam2e[256], const creal_T exp_plus_omega[1536],
        const int32_T inembed, creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_re;
    dc1_re = 0.0 * exp_plus_omega[inembed + 3 * (k + 256)].re -
             exp_plus_omega[inembed + 3 * (k + 256)].im;
    dc1_im = 0.0 * exp_plus_omega[inembed + 3 * (k + 256)].im +
             exp_plus_omega[inembed + 3 * (k + 256)].re;
    d = gam2e[k];
    d1 = gam3e[k];
    gam2e_re = d * x[k].re + d1 * Psie[k].re;
    d = d * x[k].im + d1 * Psie[k].im;
    k_e[k + 256].re = dc1_re * gam2e_re - dc1_im * d;
    k_e[k + 256].im = dc1_re * d + dc1_im * gam2e_re;
  }
}

static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T line)
{
  if (errCode != hipSuccess) {
    b_raiseCudaError(errCode, file, line, hipGetErrorName(errCode),
                     hipGetErrorString(errCode));
  }
}

static void cufftCheck(hipfftResult_t errCode, const char_T *file, uint32_T line)
{
  const char *errName;
  const char *errString;
  if (errCode != HIPFFT_SUCCESS) {
    cufftGetErrorName(static_cast<int32_T>(errCode), &errName);
    cufftGetErrorString(static_cast<int32_T>(errCode), &errString);
    raiseCudaError(errCode, file, line, errName, errString);
  }
}

static __global__ __launch_bounds__(32, 1) void d_Chi23_Runge_IntegrationSteps_(
    const creal_T exp_plus_omega[1536], creal_T *b_exp_plus_omega)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *b_exp_plus_omega = exp_plus_omega[0];
  }
}

static __global__
    __launch_bounds__(512, 1) void db_Chi23_Runge_IntegrationSteps(
        const creal_T F_e[512], const creal_T k_e[512], const int32_T inembed,
        creal_T E_temp[512], creal_T d[1536])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    d[inembed + 3 * k] = k_e[k];
    E_temp[k] = F_e[k];
  }
}

static __global__ __launch_bounds__(
    512, 1) void e_Chi23_Runge_IntegrationSteps_(const creal_T d[1536],
                                                 creal_T b_d[1536])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 1536) {
    b_d[k] = d[k];
  }
}

static __global__
    __launch_bounds__(512, 1) void eb_Chi23_Runge_IntegrationSteps(
        const creal_T d[1536], const int32_T inembed, const real_T dt,
        const creal_T F_e[512], const real_T mode_range[512],
        creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T b_d;
    b_d = mode_range[k];
    E_temp[k].re = F_e[static_cast<int32_T>(b_d) - 1].re +
                   dt * d[inembed + 3 * (static_cast<int32_T>(b_d) - 1)].re;
    E_temp[k].im = F_e[static_cast<int32_T>(b_d) - 1].im +
                   dt * d[inembed + 3 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__ __launch_bounds__(
    512, 1) void f_Chi23_Runge_IntegrationSteps_(const creal_T F_e[512],
                                                 creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    //     coder.gpu.kernelfun;
    //    coder.gpu.kernel;
    E_temp[k] = F_e[k];
  }
}

static __global__ __launch_bounds__(
    512, 1) void fb_Chi23_Runge_IntegrationSteps(const creal_T E_temp[512],
                                                 const real_T mode_range[512],
                                                 creal_T F_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    F_e[static_cast<int32_T>(mode_range[k]) - 1] = E_temp[k];
  }
}

static __global__
    __launch_bounds__(256, 1) void g_Chi23_Runge_IntegrationSteps_(
        const creal_T F_e[512], const creal_T exp_minus_omega[1536],
        creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    d = exp_minus_omega[3 * k].re;
    d1 = exp_minus_omega[3 * k].im;
    Psio[k].re = F_e[k].re * d - F_e[k].im * d1;
    Psio[k].im = F_e[k].re * d1 + F_e[k].im * d;
  }
}

static __global__ __launch_bounds__(
    512, 1) void gb_Chi23_Runge_IntegrationSteps(const creal_T F_e[512],
                                                 const creal_T shift_back[512],
                                                 const real_T mode_range[512],
                                                 creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T d;
    d = mode_range[k];
    E_temp[k].re = shift_back[static_cast<int32_T>(d) - 1].re *
                       F_e[static_cast<int32_T>(d) - 1].re -
                   shift_back[static_cast<int32_T>(d) - 1].im *
                       F_e[static_cast<int32_T>(d) - 1].im;
    E_temp[k].im = shift_back[static_cast<int32_T>(d) - 1].re *
                       F_e[static_cast<int32_T>(d) - 1].im +
                   shift_back[static_cast<int32_T>(d) - 1].im *
                       F_e[static_cast<int32_T>(d) - 1].re;
  }
}

static __global__
    __launch_bounds__(256, 1) void h_Chi23_Runge_IntegrationSteps_(
        const creal_T F_e[512], const creal_T exp_minus_omega[1536],
        creal_T Psie[256], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psio[k].re;
    dc1_re = Psio[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psio[k].re = gam2e_re;
    Psio[k].im = gam2e_im;
    gam2e_im = exp_minus_omega[3 * (k + 256)].re;
    dc1_re = exp_minus_omega[3 * (k + 256)].im;
    Psie[k].re = F_e[k + 256].re * gam2e_im - F_e[k + 256].im * dc1_re;
    Psie[k].im = F_e[k + 256].re * dc1_re + F_e[k + 256].im * gam2e_im;
  }
}

static __global__ __launch_bounds__(
    512, 1) void hb_Chi23_Runge_IntegrationSteps(const creal_T E_temp[512],
                                                 const real_T mode_range[512],
                                                 creal_T F_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    F_e[static_cast<int32_T>(mode_range[k]) - 1] = E_temp[k];
  }
}

static __global__
    __launch_bounds__(256,
                      1) void i_Chi23_Runge_IntegrationSteps_(creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psie[k].re;
    dc1_re = Psie[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psie[k].re = gam2e_re;
    Psie[k].im = gam2e_im;
  }
}

static __global__
    __launch_bounds__(512, 1) void j_Chi23_Runge_IntegrationSteps_(
        const creal_T *exp_plus_omega, const real_T H_f[512], creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T d;
    d = H_f[k];
    k_e[k].re = d * exp_plus_omega->re;
    k_e[k].im = d * exp_plus_omega->im;
  }
}

static __global__
    __launch_bounds__(256, 1) void k_Chi23_Runge_IntegrationSteps_(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_im;
    dc1_re = Psio[k].re;
    gam2e_im = -Psio[k].im;
    d = Psie[k].re;
    d1 = Psie[k].im;
    x[k].re = dc1_re * d - gam2e_im * d1;
    x[k].im = dc1_re * d1 + gam2e_im * d;
  }
}

static __global__
    __launch_bounds__(256, 1) void l_Chi23_Runge_IntegrationSteps_(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = Psio[k].re;
    d1 = Psio[k].im;
    gam2e_re = hypot(d, d1);
    dc1_re = gam2e_re * gam2e_re;
    gam2e_re = hypot(Psie[k].re, Psie[k].im);
    x[k].re = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d;
    x[k].im = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d1;
  }
}

static __global__
    __launch_bounds__(256, 1) void m_Chi23_Runge_IntegrationSteps_(
        const creal_T Psio[256], const creal_T x[256], const real_T gam3o[256],
        const real_T gam2o[256], const creal_T exp_plus_omega[1536],
        creal_T k_e[512], creal_T b_x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_im;
    //
    d = exp_plus_omega[3 * k].re;
    d1 = exp_plus_omega[3 * k].im;
    dc1_re = 0.0 * d - d1;
    dc1_im = 0.0 * d1 + d;
    d = gam2o[k];
    d1 = gam3o[k];
    gam2e_im = d * b_x[k].re + d1 * x[k].re;
    d = d * b_x[k].im + d1 * x[k].im;
    k_e[k].re = dc1_re * gam2e_im - dc1_im * d;
    k_e[k].im = dc1_re * d + dc1_im * gam2e_im;
    d = Psio[k].re;
    d1 = Psio[k].im;
    b_x[k].re = d * d - d1 * d1;
    b_x[k].im = d * d1 + d1 * d;
  }
}

static __global__ __launch_bounds__(
    256, 1) void n_Chi23_Runge_IntegrationSteps_(const creal_T Psio[256],
                                                 creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = hypot(Psio[k].re, Psio[k].im);
    d1 = d * d;
    gam2e_re = Psie[k].re;
    dc1_re = Psie[k].im;
    d = hypot(gam2e_re, dc1_re);
    gam2e_re *= 2.0 * d1 + d * d;
    Psie[k].re = gam2e_re;
    dc1_re *= 2.0 * d1 + d * d;
    Psie[k].im = dc1_re;
  }
}

static __global__
    __launch_bounds__(256, 1) void o_Chi23_Runge_IntegrationSteps_(
        const creal_T Psie[256], const creal_T x[256], const real_T gam3e[256],
        const real_T gam2e[256], const creal_T exp_plus_omega[1536],
        creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_im;
    real_T dc1_re;
    real_T gam2e_re;
    d = exp_plus_omega[3 * (k + 256)].re;
    d1 = exp_plus_omega[3 * (k + 256)].im;
    dc1_re = 0.0 * d - d1;
    dc1_im = 0.0 * d1 + d;
    d = gam2e[k];
    d1 = gam3e[k];
    gam2e_re = d * x[k].re + d1 * Psie[k].re;
    d = d * x[k].im + d1 * Psie[k].im;
    k_e[k + 256].re = dc1_re * gam2e_re - dc1_im * d;
    k_e[k + 256].im = dc1_re * d + dc1_im * gam2e_re;
  }
}

static __global__ __launch_bounds__(
    512, 1) void p_Chi23_Runge_IntegrationSteps_(const creal_T k_e[512],
                                                 creal_T d[1536])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    //
    //     k_e(1)      = k_e(1) + ;
    d[3 * k] = k_e[k];
  }
}

static __global__
    __launch_bounds__(512, 1) void q_Chi23_Runge_IntegrationSteps_(
        const creal_T d[1536], const int32_T inembed, const real_T dt,
        const creal_T E_temp[512], const real_T mode_range[512],
        creal_T b_E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T b_d;
    b_d = mode_range[k];
    b_E_temp[k].re = E_temp[static_cast<int32_T>(b_d) - 1].re +
                     dt * d[inembed + 3 * (static_cast<int32_T>(b_d) - 1)].re;
    b_E_temp[k].im = E_temp[static_cast<int32_T>(b_d) - 1].im +
                     dt * d[inembed + 3 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__ __launch_bounds__(
    512, 1) void r_Chi23_Runge_IntegrationSteps_(const creal_T E_temp[512],
                                                 const real_T mode_range[512],
                                                 creal_T b_E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    b_E_temp[static_cast<int32_T>(mode_range[k]) - 1] = E_temp[k];
  }
}

static void raiseCudaError(uint32_T errCode, const char_T *file, uint32_T line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint32_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = (char_T *)std::calloc(len + 1U, 1U);
  fn = (char_T *)std::calloc(len + 1U, 1U);
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(errCode, (char_T *)errorName, (char_T *)errorString, &rtInfo,
                 emlrtRootTLSGlobal);
}

static __global__
    __launch_bounds__(256, 1) void s_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_minus_omega[1536], const int32_T inembed,
        const creal_T E_temp[512], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    d = E_temp[k].re;
    d1 = E_temp[k].im;
    Psio[k].re = d * exp_minus_omega[inembed + 3 * k].re -
                 d1 * exp_minus_omega[inembed + 3 * k].im;
    Psio[k].im = d * exp_minus_omega[inembed + 3 * k].im +
                 d1 * exp_minus_omega[inembed + 3 * k].re;
  }
}

static __global__
    __launch_bounds__(256, 1) void t_Chi23_Runge_IntegrationSteps_(
        const creal_T exp_minus_omega[1536], const int32_T inembed,
        const creal_T E_temp[512], creal_T Psie[256], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psio[k].re;
    dc1_re = Psio[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psio[k].re = gam2e_re;
    Psio[k].im = gam2e_im;
    gam2e_im = E_temp[k + 256].re;
    dc1_re = E_temp[k + 256].im;
    Psie[k].re = gam2e_im * exp_minus_omega[inembed + 3 * (k + 256)].re -
                 dc1_re * exp_minus_omega[inembed + 3 * (k + 256)].im;
    Psie[k].im = gam2e_im * exp_minus_omega[inembed + 3 * (k + 256)].im +
                 dc1_re * exp_minus_omega[inembed + 3 * (k + 256)].re;
  }
}

static __global__
    __launch_bounds__(256,
                      1) void u_Chi23_Runge_IntegrationSteps_(creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T dc1_re;
    real_T gam2e_im;
    real_T gam2e_re;
    gam2e_im = Psie[k].re;
    dc1_re = Psie[k].im;
    if (dc1_re == 0.0) {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = 0.0;
    } else if (gam2e_im == 0.0) {
      gam2e_re = 0.0;
      gam2e_im = dc1_re / 256.0;
    } else {
      gam2e_re = gam2e_im / 256.0;
      gam2e_im = dc1_re / 256.0;
    }
    Psie[k].re = gam2e_re;
    Psie[k].im = gam2e_im;
  }
}

static __global__ __launch_bounds__(32, 1) void v_Chi23_Runge_IntegrationSteps_(
    const creal_T exp_plus_omega[1536], const int32_T i2,
    creal_T *b_exp_plus_omega)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *b_exp_plus_omega = exp_plus_omega[i2 + 1];
  }
}

static __global__
    __launch_bounds__(512, 1) void w_Chi23_Runge_IntegrationSteps_(
        const creal_T *exp_plus_omega, const real_T H_f[512], creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T d;
    d = H_f[k];
    k_e[k].re = d * exp_plus_omega->re;
    k_e[k].im = d * exp_plus_omega->im;
  }
}

static __global__
    __launch_bounds__(256, 1) void x_Chi23_Runge_IntegrationSteps_(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_im;
    dc1_re = Psio[k].re;
    gam2e_im = -Psio[k].im;
    d = Psie[k].re;
    d1 = Psie[k].im;
    x[k].re = dc1_re * d - gam2e_im * d1;
    x[k].im = dc1_re * d1 + gam2e_im * d;
  }
}

static __global__
    __launch_bounds__(256, 1) void y_Chi23_Runge_IntegrationSteps_(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T dc1_re;
    real_T gam2e_re;
    d = Psio[k].re;
    d1 = Psio[k].im;
    gam2e_re = hypot(d, d1);
    dc1_re = gam2e_re * gam2e_re;
    gam2e_re = hypot(Psie[k].re, Psie[k].im);
    x[k].re = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d;
    x[k].im = (dc1_re + 2.0 * (gam2e_re * gam2e_re)) * d1;
  }
}

void Chi23_Runge_IntegrationSteps(
    real_T nt, real_T dt, const creal_T d[1536], creal_T F_e[512],
    const creal_T exp_plus_omega[1536], const creal_T exp_minus_omega[1536],
    const struct0_T *Runge, real_T t, const creal_T shift_back[512],
    const real_T gam2o[256], const real_T gam2e[256], const real_T gam3o[256],
    const real_T gam3e[256], const real_T H_f[512],
    const real_T mode_range[512])
{
  creal_T(*gpu_d)[1536];
  creal_T(*b_gpu_E_temp)[512];
  creal_T(*gpu_E_temp)[512];
  creal_T(*gpu_k_e)[512];
  creal_T(*b_gpu_x)[256];
  creal_T(*gpu_Psie)[256];
  creal_T(*gpu_Psio)[256];
  creal_T(*gpu_x)[256];
  creal_T *b_gpu_exp_plus_omega;
  creal_T *gpu_exp_plus_omega;
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T i3;
  (void)t;
#define CUDACHECK(errCall) checkCudaError(errCall, __FILE__, __LINE__)
  checkCudaError(hipGetLastError(), __FILE__, __LINE__);
  hipMalloc(&b_gpu_exp_plus_omega, 16ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&b_gpu_E_temp, 8192ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&b_gpu_x, 4096ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_x, 4096ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_k_e, 8192ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_Psie, 4096ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_Psio, 4096ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_E_temp, 8192ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_d, 24576ULL);
  CUDACHECK(hipGetLastError());
  hipMalloc(&gpu_exp_plus_omega, 16ULL);
  CUDACHECK(hipGetLastError());
  i = static_cast<int32_T>(nt);
  if (0 <= static_cast<int32_T>(nt) - 1) {
    d_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        exp_plus_omega, gpu_exp_plus_omega);
    CUDACHECK(hipGetLastError());
    i1 = static_cast<int32_T>(Runge->s + -1.0);
    i2 = static_cast<int32_T>(Runge->s);
  }
  for (int32_T ni{0}; ni < i; ni++) {
    hipfftHandle b_fftPlanHandle;
    hipfftHandle c_fftPlanHandle;
    hipfftHandle d_fftPlanHandle;
    hipfftHandle e_fftPlanHandle;
    hipfftHandle f_fftPlanHandle;
    hipfftHandle fftPlanHandle;
    real_T b_dt;
    int32_T inembed;
    e_Chi23_Runge_IntegrationSteps_<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        d, *gpu_d);
    CUDACHECK(hipGetLastError());
    //     coder.gpu.kernelfun;
    //    coder.gpu.kernel;
    f_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        F_e, *gpu_E_temp);
    CUDACHECK(hipGetLastError());
    g_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        F_e, exp_minus_omega, *gpu_Psio);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(
        hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psio)[0],
                     (hipfftDoubleComplex *)&(*gpu_Psio)[0], HIPFFT_BACKWARD),
        __FILE__, __LINE__);
    h_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        F_e, exp_minus_omega, *gpu_Psie, *gpu_Psio);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    b_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(
        hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                     (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_BACKWARD),
        __FILE__, __LINE__);
    i_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie);
    CUDACHECK(hipGetLastError());
    j_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        gpu_exp_plus_omega, H_f, *gpu_k_e);
    CUDACHECK(hipGetLastError());
    k_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_Psio, *gpu_x);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    c_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(hipfftExecZ2Z(c_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                            (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD),
               __FILE__, __LINE__);
    l_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_Psio, *b_gpu_x);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    d_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(
        hipfftExecZ2Z(d_fftPlanHandle, (hipfftDoubleComplex *)&(*b_gpu_x)[0],
                     (hipfftDoubleComplex *)&(*b_gpu_x)[0], HIPFFT_FORWARD),
        __FILE__, __LINE__);
    //
    m_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psio, *b_gpu_x, gam3o, gam2o, exp_plus_omega, *gpu_k_e, *gpu_x);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    e_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(hipfftExecZ2Z(e_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                            (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD),
               __FILE__, __LINE__);
    n_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psio, *gpu_Psie);
    CUDACHECK(hipGetLastError());
    inembed = 256;
    f_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
    cufftCheck(
        hipfftExecZ2Z(f_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                     (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_FORWARD),
        __FILE__, __LINE__);
    o_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_x, gam3e, gam2e, exp_plus_omega, *gpu_k_e);
    CUDACHECK(hipGetLastError());
    //
    //     k_e(1)      = k_e(1) + ;
    p_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_k_e, *gpu_d);
    CUDACHECK(hipGetLastError());
    if (0 <= i1 - 1) {
      i3 = static_cast<int32_T>(Runge->s - 1.0);
    }
    for (int32_T b_i2{0}; b_i2 < i1; b_i2++) {
      for (inembed = 0; inembed < i3; inembed++) {
        b_dt = Runge->b[(b_i2 + 3 * inembed) + 1];
        if (b_dt != 0.0) {
          b_dt *= dt;
          q_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              *gpu_d, inembed, b_dt, *gpu_E_temp, mode_range, *b_gpu_E_temp);
          CUDACHECK(hipGetLastError());
          r_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              *b_gpu_E_temp, mode_range, *gpu_E_temp);
          CUDACHECK(hipGetLastError());
        }
      }
      hipfftHandle g_fftPlanHandle;
      hipfftHandle h_fftPlanHandle;
      hipfftHandle i_fftPlanHandle;
      hipfftHandle j_fftPlanHandle;
      hipfftHandle k_fftPlanHandle;
      hipfftHandle l_fftPlanHandle;
      s_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          exp_minus_omega, b_i2 + 1, *gpu_E_temp, *gpu_Psio);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      g_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(g_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psio)[0],
                       (hipfftDoubleComplex *)&(*gpu_Psio)[0], HIPFFT_BACKWARD),
          __FILE__, __LINE__);
      t_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          exp_minus_omega, b_i2 + 1, *gpu_E_temp, *gpu_Psie, *gpu_Psio);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      h_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(h_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                       (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_BACKWARD),
          __FILE__, __LINE__);
      u_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie);
      CUDACHECK(hipGetLastError());
      v_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          exp_plus_omega, b_i2, b_gpu_exp_plus_omega);
      CUDACHECK(hipGetLastError());
      w_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          b_gpu_exp_plus_omega, H_f, *gpu_k_e);
      CUDACHECK(hipGetLastError());
      x_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_Psio, *gpu_x);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      i_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(i_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                       (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD),
          __FILE__, __LINE__);
      y_Chi23_Runge_IntegrationSteps_<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_Psio, *b_gpu_x);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      j_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(j_fftPlanHandle, (hipfftDoubleComplex *)&(*b_gpu_x)[0],
                       (hipfftDoubleComplex *)&(*b_gpu_x)[0], HIPFFT_FORWARD),
          __FILE__, __LINE__);
      //
      ab_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psio, *b_gpu_x, gam3o, gam2o, exp_plus_omega, b_i2 + 1, *gpu_k_e,
          *gpu_x);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      k_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(k_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                       (hipfftDoubleComplex *)&(*gpu_x)[0], HIPFFT_FORWARD),
          __FILE__, __LINE__);
      bb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psio, *gpu_Psie);
      CUDACHECK(hipGetLastError());
      inembed = 256;
      l_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, HIPFFT_Z2Z, 1);
      cufftCheck(
          hipfftExecZ2Z(l_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                       (hipfftDoubleComplex *)&(*gpu_Psie)[0], HIPFFT_FORWARD),
          __FILE__, __LINE__);
      cb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_x, gam3e, gam2e, exp_plus_omega, b_i2 + 1, *gpu_k_e);
      CUDACHECK(hipGetLastError());
      //
      //     k_e(1)      = k_e(1) + ;
      db_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          F_e, *gpu_k_e, b_i2 + 1, *gpu_E_temp, *gpu_d);
      CUDACHECK(hipGetLastError());
    }
    for (inembed = 0; inembed < i2; inembed++) {
      b_dt = Runge->c[inembed];
      if (b_dt != 0.0) {
        b_dt *= dt;
        eb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            *gpu_d, inembed, b_dt, F_e, mode_range, *b_gpu_E_temp);
        CUDACHECK(hipGetLastError());
        fb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            *b_gpu_E_temp, mode_range, F_e);
        CUDACHECK(hipGetLastError());
      }
    }
    gb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        F_e, shift_back, mode_range, *b_gpu_E_temp);
    CUDACHECK(hipGetLastError());
    hb_Chi23_Runge_IntegrationSteps<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *b_gpu_E_temp, mode_range, F_e);
    CUDACHECK(hipGetLastError());
    // ,Runge,exp_plus_omega,exp_minus_omega,Temp,shift_back
  }
  hipFree(gpu_exp_plus_omega);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_d);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_E_temp);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_Psio);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_Psie);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_k_e);
  CUDACHECK(hipGetLastError());
  hipFree(*gpu_x);
  CUDACHECK(hipGetLastError());
  hipFree(*b_gpu_x);
  CUDACHECK(hipGetLastError());
  hipFree(*b_gpu_E_temp);
  CUDACHECK(hipGetLastError());
  hipFree(b_gpu_exp_plus_omega);
  CUDACHECK(hipGetLastError());
#undef CUDACHECK
}

// End of code generation (Chi23_Runge_IntegrationSteps.cu)

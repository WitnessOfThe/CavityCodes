#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_Chi23_Runge_Kuarong_GPU_api.cu
//
// Code generation for function '_coder_Chi23_Runge_Kuarong_GPU_api'
//

// Include files
#include "_coder_Chi23_Runge_Kuarong_GPU_api.h"
#include "Chi23_Runge_Kuarong_GPU.h"
#include "Chi23_Runge_Kuarong_GPU_data.h"
#include "Chi23_Runge_Kuarong_GPU_emxutil.h"
#include "Chi23_Runge_Kuarong_GPU_types.h"
#include "rt_nonfinite.h"

// Function Declarations
static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               creal_T y[256]);

static struct1_T b_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId);

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[12]);

static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[1032]);

static void c_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[512]);

static void c_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[2]);

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId);

static void d_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[256]);

static struct4_T d_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId);

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[256]);

static void e_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[512]);

static creal_T e_emlrt_marshallIn(const mxArray *u,
                                  const emlrtMsgIdentifier *parentId);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct0_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[12]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct2_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct3_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[656]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct5_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct6_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             creal_T y[512]);

static void emlrt_marshallIn(const mxArray *Runge, const char_T *identifier,
                             struct7_T *y);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct7_T *y);

static void emlrt_marshallIn(const mxArray *Temp, const char_T *identifier,
                             struct0_T *y);

static const mxArray *emlrt_marshallOut(const struct8_T u);

static void f_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId);

static void f_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[9]);

static void g_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[3]);

static real_T g_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

static void h_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[656]);

static creal_T h_emlrt_marshallIn(const mxArray *src,
                                  const emlrtMsgIdentifier *msgId);

static void i_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[1032]);

static void j_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[2]);

static void k_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[256]);

static void l_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[512]);

static void m_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[9]);

static void n_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[3]);

// Function Definitions
static struct1_T b_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[6]{"Runge_Type", "dt", "s_t",
                                     "T",          "dd", "CW_num"};
  emlrtMsgIdentifier thisId;
  struct1_T y;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 6,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "Runge_Type";
  emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                     (const char_T *)"Runge_Type")),
      &thisId, y.Runge_Type);
  thisId.fIdentifier = "dt";
  y.dt = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"dt")),
      &thisId);
  thisId.fIdentifier = "s_t";
  y.s_t = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                     (const char_T *)"s_t")),
      &thisId);
  thisId.fIdentifier = "T";
  y.T =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 3, (const char_T *)"T")),
                         &thisId);
  thisId.fIdentifier = "dd";
  y.dd = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                     (const char_T *)"dd")),
      &thisId);
  thisId.fIdentifier = "CW_num";
  y.CW_num = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                     (const char_T *)"CW_num")),
      &thisId);
  emlrtDestroyArray(&u);
  return y;
}

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, char_T ret[12])
{
  static const int32_T dims[2]{1, 12};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"char", false, 2U, (void *)&dims[0]);
  emlrtImportCharArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 12);
  emlrtDestroyArray(&src);
}

static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[1032])
{
  i_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void b_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               creal_T y[256])
{
  d_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void c_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[2])
{
  j_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void c_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[512])
{
  static const int32_T dims[2]{1, 512};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", true, 2U, (void *)&dims[0]);
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 8, true);
  emlrtDestroyArray(&src);
}

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = g_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static void d_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               creal_T ret[256])
{
  static const int32_T dims[2]{1, 256};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", true, 2U, (void *)&dims[0]);
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret[0], 8, true);
  emlrtDestroyArray(&src);
}

static struct4_T d_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[7]{"eta", "delta", "m_p", "W",
                                     "N",   "Nenv",  "g"};
  emlrtMsgIdentifier thisId;
  struct4_T y;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 7,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "eta";
  y.eta = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                     (const char_T *)"eta")),
      &thisId);
  thisId.fIdentifier = "delta";
  y.delta = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"delta")),
      &thisId);
  thisId.fIdentifier = "m_p";
  y.m_p = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                     (const char_T *)"m_p")),
      &thisId);
  thisId.fIdentifier = "W";
  y.W =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 3, (const char_T *)"W")),
                         &thisId);
  thisId.fIdentifier = "N";
  y.N =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 4, (const char_T *)"N")),
                         &thisId);
  thisId.fIdentifier = "Nenv";
  y.Nenv = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                     (const char_T *)"Nenv")),
      &thisId);
  thisId.fIdentifier = "g";
  y.g =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 6, (const char_T *)"g")),
                         &thisId);
  emlrtDestroyArray(&u);
  return y;
}

static void d_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[256])
{
  k_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void e_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId,
                               real_T y[512])
{
  l_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static creal_T e_emlrt_marshallIn(const mxArray *u,
                                  const emlrtMsgIdentifier *parentId)
{
  creal_T y;
  y = h_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct5_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[6]{"N", "lenght", "dphi", "phi", "dk", "k"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 6,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "N";
  y->N =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 0, (const char_T *)"N")),
                         &thisId);
  thisId.fIdentifier = "lenght";
  y->lenght = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"lenght")),
      &thisId);
  thisId.fIdentifier = "dphi";
  y->dphi = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                     (const char_T *)"dphi")),
      &thisId);
  thisId.fIdentifier = "phi";
  d_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                                    (const char_T *)"phi")),
                     &thisId, y->phi);
  thisId.fIdentifier = "dk";
  y->dk = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                     (const char_T *)"dk")),
      &thisId);
  thisId.fIdentifier = "k";
  d_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                                    (const char_T *)"k")),
                     &thisId, y->k);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *Temp, const char_T *identifier,
                             struct0_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(Temp), &thisId, y);
  emlrtDestroyArray(&Temp);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct0_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[5]{"Met", "Par", "In", "Space", "Eq"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 5,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "Met";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                                  (const char_T *)"Met")),
                   &thisId);
  thisId.fIdentifier = "Par";
  y->Par = b_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"Par")),
      &thisId);
  thisId.fIdentifier = "In";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                                  (const char_T *)"In")),
                   &thisId, &y->In);
  thisId.fIdentifier = "Space";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                                  (const char_T *)"Space")),
                   &thisId, &y->Space);
  thisId.fIdentifier = "Eq";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                                  (const char_T *)"Eq")),
                   &thisId, &y->Eq);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId)
{
  f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, char_T y[12])
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct2_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[33]{
      "eta",        "delta_o",   "W",        "ko",     "ke",      "eps",
      "eps_2",      "Do",        "De",       "gam2o",  "gam2e",   "gam3o",
      "gam3e",      "Finess",    "Finess_e", "H_Star", "Wf_Star", "Ws_Star",
      "Omega_Star", "D",         "N_mode",   "N",      "kMI",     "mu_bl",
      "delta",      "delta_e",   "H_f",      "H_s",    "Omega_f", "Omega_s",
      "d",          "Psi_Start", "t_start"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 33,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "eta";
  y->eta = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                     (const char_T *)"eta")),
      &thisId);
  thisId.fIdentifier = "delta_o";
  y->delta_o = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"delta_o")),
      &thisId);
  thisId.fIdentifier = "W";
  y->W =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 2, (const char_T *)"W")),
                         &thisId);
  thisId.fIdentifier = "ko";
  y->ko = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                     (const char_T *)"ko")),
      &thisId);
  thisId.fIdentifier = "ke";
  y->ke = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                     (const char_T *)"ke")),
      &thisId);
  thisId.fIdentifier = "eps";
  y->eps = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                     (const char_T *)"eps")),
      &thisId);
  thisId.fIdentifier = "eps_2";
  y->eps_2 = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 6,
                                     (const char_T *)"eps_2")),
      &thisId);
  thisId.fIdentifier = "Do";
  y->Do = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 7,
                                     (const char_T *)"Do")),
      &thisId);
  thisId.fIdentifier = "De";
  y->De = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 8,
                                     (const char_T *)"De")),
      &thisId);
  thisId.fIdentifier = "gam2o";
  y->gam2o = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 9,
                                     (const char_T *)"gam2o")),
      &thisId);
  thisId.fIdentifier = "gam2e";
  y->gam2e = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 10,
                                     (const char_T *)"gam2e")),
      &thisId);
  thisId.fIdentifier = "gam3o";
  y->gam3o = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 11,
                                     (const char_T *)"gam3o")),
      &thisId);
  thisId.fIdentifier = "gam3e";
  y->gam3e = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 12,
                                     (const char_T *)"gam3e")),
      &thisId);
  thisId.fIdentifier = "Finess";
  y->Finess = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 13,
                                     (const char_T *)"Finess")),
      &thisId);
  thisId.fIdentifier = "Finess_e";
  y->Finess_e = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 14,
                                     (const char_T *)"Finess_e")),
      &thisId);
  thisId.fIdentifier = "H_Star";
  y->H_Star = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 15,
                                     (const char_T *)"H_Star")),
      &thisId);
  thisId.fIdentifier = "Wf_Star";
  y->Wf_Star = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 16,
                                     (const char_T *)"Wf_Star")),
      &thisId);
  thisId.fIdentifier = "Ws_Star";
  y->Ws_Star = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 17,
                                     (const char_T *)"Ws_Star")),
      &thisId);
  thisId.fIdentifier = "Omega_Star";
  y->Omega_Star = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 18,
                                     (const char_T *)"Omega_Star")),
      &thisId);
  thisId.fIdentifier = "D";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 19,
                                                  (const char_T *)"D")),
                   &thisId, &y->D);
  thisId.fIdentifier = "N_mode";
  y->N_mode = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 20,
                                     (const char_T *)"N_mode")),
      &thisId);
  thisId.fIdentifier = "N";
  y->N = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 21,
                                     (const char_T *)"N")),
      &thisId);
  thisId.fIdentifier = "kMI";
  c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                    22, (const char_T *)"kMI")),
                     &thisId, y->kMI);
  thisId.fIdentifier = "mu_bl";
  y->mu_bl = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 23,
                                     (const char_T *)"mu_bl")),
      &thisId);
  thisId.fIdentifier = "delta";
  y->delta = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 24,
                                     (const char_T *)"delta")),
      &thisId);
  thisId.fIdentifier = "delta_e";
  y->delta_e = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 25,
                                     (const char_T *)"delta_e")),
      &thisId);
  thisId.fIdentifier = "H_f";
  y->H_f = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 26,
                                     (const char_T *)"H_f")),
      &thisId);
  thisId.fIdentifier = "H_s";
  y->H_s = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 27,
                                     (const char_T *)"H_s")),
      &thisId);
  thisId.fIdentifier = "Omega_f";
  y->Omega_f = e_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 28,
                                     (const char_T *)"Omega_f")),
      &thisId);
  thisId.fIdentifier = "Omega_s";
  y->Omega_s = e_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 29,
                                     (const char_T *)"Omega_s")),
      &thisId);
  thisId.fIdentifier = "d";
  y->d = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 30,
                                     (const char_T *)"d")),
      &thisId);
  thisId.fIdentifier = "Psi_Start";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 31,
                                                  (const char_T *)"Psi_Start")),
                   &thisId, y->Psi_Start);
  thisId.fIdentifier = "t_start";
  y->t_start = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 32,
                                     (const char_T *)"t_start")),
      &thisId);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct3_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[12]{
      "omega",  "m", "D1",        "n_o",        "n_e",   "gamma2",
      "gamma3", "S", "gamma2_SH", "gamma2_OPO", "kappa", "In"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 12,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "omega";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                                  (const char_T *)"omega")),
                   &thisId, y->omega);
  thisId.fIdentifier = "m";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                                  (const char_T *)"m")),
                   &thisId, y->m);
  thisId.fIdentifier = "D1";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                                  (const char_T *)"D1")),
                   &thisId, y->D1);
  thisId.fIdentifier = "n_o";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                                    (const char_T *)"n_o")),
                     &thisId, y->n_o);
  thisId.fIdentifier = "n_e";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                                    (const char_T *)"n_e")),
                     &thisId, y->n_e);
  thisId.fIdentifier = "gamma2";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                                  (const char_T *)"gamma2")),
                   &thisId, y->gamma2);
  thisId.fIdentifier = "gamma3";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 6,
                                                  (const char_T *)"gamma3")),
                   &thisId, y->gamma3);
  thisId.fIdentifier = "S";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 7,
                                                    (const char_T *)"S")),
                     &thisId, y->S);
  thisId.fIdentifier = "gamma2_SH";
  b_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 8,
                                     (const char_T *)"gamma2_SH")),
      &thisId, y->gamma2_SH);
  thisId.fIdentifier = "gamma2_OPO";
  b_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 9,
                                     (const char_T *)"gamma2_OPO")),
      &thisId, y->gamma2_OPO);
  thisId.fIdentifier = "kappa";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 10,
                                                  (const char_T *)"kappa")),
                   &thisId, y->kappa);
  thisId.fIdentifier = "In";
  y->In = d_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 11,
                                     (const char_T *)"In")),
      &thisId);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[656])
{
  h_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct6_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[19]{
      "H_f",   "H_s",        "delta_o", "delta_e", "eps", "ko", "ke",
      "gam2o", "gam2e",      "gam3o",   "gam3e",   "Do",  "De", "d",
      "norm",  "mode_range", "Lo",      "Le",      "L"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 19,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "H_f";
  y->H_f = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                     (const char_T *)"H_f")),
      &thisId);
  thisId.fIdentifier = "H_s";
  y->H_s = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                     (const char_T *)"H_s")),
      &thisId);
  thisId.fIdentifier = "delta_o";
  y->delta_o = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                     (const char_T *)"delta_o")),
      &thisId);
  thisId.fIdentifier = "delta_e";
  y->delta_e = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 3,
                                     (const char_T *)"delta_e")),
      &thisId);
  thisId.fIdentifier = "eps";
  y->eps = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 4,
                                     (const char_T *)"eps")),
      &thisId);
  thisId.fIdentifier = "ko";
  y->ko = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 5,
                                     (const char_T *)"ko")),
      &thisId);
  thisId.fIdentifier = "ke";
  y->ke = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 6,
                                     (const char_T *)"ke")),
      &thisId);
  thisId.fIdentifier = "gam2o";
  d_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 7,
                                                    (const char_T *)"gam2o")),
                     &thisId, y->gam2o);
  thisId.fIdentifier = "gam2e";
  d_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 8,
                                                    (const char_T *)"gam2e")),
                     &thisId, y->gam2e);
  thisId.fIdentifier = "gam3o";
  d_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 9,
                                                    (const char_T *)"gam3o")),
                     &thisId, y->gam3o);
  thisId.fIdentifier = "gam3e";
  d_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 10,
                                     (const char_T *)"gam3e")),
      &thisId, y->gam3e);
  thisId.fIdentifier = "Do";
  y->Do = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 11,
                                     (const char_T *)"Do")),
      &thisId);
  thisId.fIdentifier = "De";
  y->De = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 12,
                                     (const char_T *)"De")),
      &thisId);
  thisId.fIdentifier = "d";
  y->d = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 13,
                                     (const char_T *)"d")),
      &thisId);
  thisId.fIdentifier = "norm";
  y->norm = c_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 14,
                                     (const char_T *)"norm")),
      &thisId);
  thisId.fIdentifier = "mode_range";
  e_emlrt_marshallIn(
      emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 15,
                                     (const char_T *)"mode_range")),
      &thisId, y->mode_range);
  thisId.fIdentifier = "Lo";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                    16, (const char_T *)"Lo")),
                     &thisId, y->Lo);
  thisId.fIdentifier = "Le";
  b_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0,
                                                    17, (const char_T *)"Le")),
                     &thisId, y->Le);
  thisId.fIdentifier = "L";
  emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 18,
                                                  (const char_T *)"L")),
                   &thisId, y->L);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, creal_T y[512])
{
  c_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *Runge, const char_T *identifier,
                             struct7_T *y)
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(Runge), &thisId, y);
  emlrtDestroyArray(&Runge);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, struct7_T *y)
{
  static const int32_T dims{0};
  static const char_T *fieldNames[4]{"b", "a", "c", "s"};
  emlrtMsgIdentifier thisId;
  thisId.fParent = parentId;
  thisId.bParentIsCell = false;
  emlrtCheckStructR2012b(emlrtRootTLSGlobal, parentId, u, 4,
                         (const char_T **)&fieldNames[0], 0U, (void *)&dims);
  thisId.fIdentifier = "b";
  f_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 0,
                                                    (const char_T *)"b")),
                     &thisId, y->b);
  thisId.fIdentifier = "a";
  g_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 1,
                                                    (const char_T *)"a")),
                     &thisId, y->a);
  thisId.fIdentifier = "c";
  g_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(emlrtRootTLSGlobal, u, 0, 2,
                                                    (const char_T *)"c")),
                     &thisId, y->c);
  thisId.fIdentifier = "s";
  y->s =
      c_emlrt_marshallIn(emlrtAlias(emlrtGetFieldR2017b(
                             emlrtRootTLSGlobal, u, 0, 3, (const char_T *)"s")),
                         &thisId);
  emlrtDestroyArray(&u);
}

static const mxArray *emlrt_marshallOut(const struct8_T u)
{
  static const char_T *sv1[3]{"Psio", "Psie", "t"};
  const mxArray *b_y;
  const mxArray *c_y;
  const mxArray *d_y;
  const mxArray *m;
  const mxArray *y;
  int32_T iv[2];
  y = nullptr;
  emlrtAssign(&y, emlrtCreateStructMatrix(1, 1, 3, (const char_T **)&sv1[0]));
  b_y = nullptr;
  iv[0] = u.Psio->size[0];
  iv[1] = u.Psio->size[1];
  m = emlrtCreateNumericArray(2, &iv[0], mxDOUBLE_CLASS, mxCOMPLEX);
  emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u.Psio->data[0], 8);
  emlrtAssign(&b_y, m);
  emlrtSetFieldR2017b(y, 0, (const char_T *)"Psio", b_y, 0);
  c_y = nullptr;
  iv[0] = u.Psie->size[0];
  iv[1] = u.Psie->size[1];
  m = emlrtCreateNumericArray(2, &iv[0], mxDOUBLE_CLASS, mxCOMPLEX);
  emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u.Psie->data[0], 8);
  emlrtAssign(&c_y, m);
  emlrtSetFieldR2017b(y, 0, (const char_T *)"Psie", c_y, 1);
  d_y = nullptr;
  iv[0] = 1;
  iv[1] = u.t->size[1];
  m = emlrtCreateNumericArray(2, &iv[0], mxDOUBLE_CLASS, mxCOMPLEX);
  emlrtExportNumericArrayR2013b(emlrtRootTLSGlobal, m, &u.t->data[0], 8);
  emlrtAssign(&d_y, m);
  emlrtSetFieldR2017b(y, 0, (const char_T *)"t", d_y, 2);
  return y;
}

static void f_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims[2]{0, 0};
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  emlrtMxGetData(src);
  emlrtDestroyArray(&src);
}

static void f_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[9])
{
  m_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void g_emlrt_marshallIn(const mxArray *u,
                               const emlrtMsgIdentifier *parentId, real_T y[3])
{
  n_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static real_T g_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 0U, (void *)&dims);
  ret = *(real_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static creal_T h_emlrt_marshallIn(const mxArray *src,
                                  const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  creal_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", true, 0U, (void *)&dims);
  emlrtImportArrayR2015b(emlrtRootTLSGlobal, src, &ret, 8, true);
  emlrtDestroyArray(&src);
  return ret;
}

static void h_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[656])
{
  static const int32_T dims[2]{1, 656};
  real_T(*r)[656];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[656])emlrtMxGetData(src);
  for (int32_T i{0}; i < 656; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static void i_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId,
                               real_T ret[1032])
{
  static const int32_T dims[2]{1, 1032};
  real_T(*r)[1032];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[1032])emlrtMxGetData(src);
  for (int32_T i{0}; i < 1032; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static void j_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[2])
{
  static const int32_T dims[2]{1, 2};
  real_T(*r)[2];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[2])emlrtMxGetData(src);
  ret[0] = (*r)[0];
  ret[1] = (*r)[1];
  emlrtDestroyArray(&src);
}

static void k_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[256])
{
  static const int32_T dims[2]{1, 256};
  real_T(*r)[256];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[256])emlrtMxGetData(src);
  for (int32_T i{0}; i < 256; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static void l_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[512])
{
  static const int32_T dims[2]{1, 512};
  real_T(*r)[512];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[512])emlrtMxGetData(src);
  for (int32_T i{0}; i < 512; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static void m_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[9])
{
  static const int32_T dims[2]{3, 3};
  real_T(*r)[9];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[9])emlrtMxGetData(src);
  for (int32_T i{0}; i < 9; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static void n_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[3])
{
  static const int32_T dims[2]{1, 3};
  real_T(*r)[3];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src,
                          (const char_T *)"double", false, 2U,
                          (void *)&dims[0]);
  r = (real_T(*)[3])emlrtMxGetData(src);
  ret[0] = (*r)[0];
  ret[1] = (*r)[1];
  ret[2] = (*r)[2];
  emlrtDestroyArray(&src);
}

void Chi23_Runge_Kuarong_GPU_api(const mxArray *const prhs[2],
                                 const mxArray *plhs[1])
{
  struct0_T Temp;
  struct7_T Runge;
  struct8_T Sol;
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInitStruct_struct8_T(&Sol, true);
  // Marshall function inputs
  emlrt_marshallIn(emlrtAliasP(prhs[0]), "Temp", &Temp);
  emlrt_marshallIn(emlrtAliasP(prhs[1]), "Runge", &Runge);
  // Invoke the target function
  Chi23_Runge_Kuarong_GPU(&Temp, &Runge, &Sol);
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(Sol);
  emxFreeStruct_struct8_T(&Sol);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
}

// End of code generation (_coder_Chi23_Runge_Kuarong_GPU_api.cu)

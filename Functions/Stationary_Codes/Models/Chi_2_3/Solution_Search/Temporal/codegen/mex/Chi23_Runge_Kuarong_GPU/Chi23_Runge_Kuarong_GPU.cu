#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi23_Runge_Kuarong_GPU.cu
//
// Code generation for function 'Chi23_Runge_Kuarong_GPU'
//

// Include files
#include "Chi23_Runge_Kuarong_GPU.h"
#include "Chi23_Runge_Kuarong_GPU_data.h"
#include "Chi23_Runge_Kuarong_GPU_emxutil.h"
#include "Chi23_Runge_Kuarong_GPU_types.h"
#include "rt_nonfinite.h"
#include "MWCUFFTPlanManager.hpp"
#include "MWCudaDimUtility.hpp"
#include "MWLaunchParametersUtilities.hpp"
#include "hipfft/hipfft.h"
#include <cmath>
#include <cstring>

// Function Declarations
static __global__ void Chi23_Runge_Kuarong_GPU_kernel1(const int32_T Runge,
                                                       emxArray_cuint8_T d);

static __global__ void
Chi23_Runge_Kuarong_GPU_kernel2(const real_T Par_dt, const struct0_T *Temp,
                                const struct7_T Runge,
                                creal_T exp_plus_omega_contents[1536],
                                creal_T exp_minus_omega_contents[1536]);

static __global__ void Chi23_Runge_Kuarong_GPU_kernel3(const struct0_T *Temp,
                                                       creal_T F_e[512],
                                                       creal_T y[512]);

static __global__ void Chi23_Runge_Kuarong_GPU_kernel4(const struct0_T *Temp,
                                                       int32_T *i);

static __global__ void
Chi23_Runge_Kuarong_GPU_kernel5(const emxArray_cuint8_T d, const int32_T b_d,
                                emxArray_creal_T c_d);

static __global__ void Chi23_Runge_Kuarong_GPU_kernel6(const creal_T F_e[512],
                                                       creal_T E_temp[512]);

static __global__ void
Chi23_Runge_Kuarong_GPU_kernel7(const creal_T exp_minus_omega_contents[1536],
                                const creal_T F_e[512], creal_T Psio[256]);

static __global__ void
Chi23_Runge_Kuarong_GPU_kernel8(const creal_T exp_minus_omega_contents[1536],
                                const creal_T F_e[512], creal_T Psie[256],
                                creal_T Psio[256]);

static __global__ void Chi23_Runge_Kuarong_GPU_kernel9(creal_T Psie[256]);

static __global__ void ab_Chi23_Runge_Kuarong_GPU_kern(const creal_T F_e[512],
                                                       const creal_T y[512],
                                                       const struct0_T *Temp,
                                                       creal_T b_y[512]);

static __global__ void b_Chi23_Runge_Kuarong_GPU_kerne(creal_T k_e[512]);

static __global__ void bb_Chi23_Runge_Kuarong_GPU_kern(const creal_T y[512],
                                                       const struct0_T *Temp,
                                                       creal_T F_e[512]);

static __global__ void c_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void d_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void
e_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psio[256], const creal_T x[256],
                                const struct0_T *Temp,
                                const creal_T exp_plus_omega_contents[1536],
                                creal_T k_e[512], creal_T b_x[256]);

static __global__ void f_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psio[256],
                                                       creal_T Psie[256]);

static __global__ void g_Chi23_Runge_Kuarong_GPU_kerne(
    const creal_T Psie[256], const creal_T x[256], const struct0_T *Temp,
    const creal_T exp_plus_omega_contents[1536], creal_T k_e[512]);

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu);

static void gpuEmxEnsureCapacity_cuint8_T(const emxArray_cuint8_T *cpu,
                                          emxArray_cuint8_T *gpu);

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu);

static void gpuEmxFree_cuint8_T(emxArray_cuint8_T *gpu);

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu);

static void gpuEmxReset_cuint8_T(emxArray_cuint8_T *gpu);

static __global__ void h_Chi23_Runge_Kuarong_GPU_kerne(const real_T im,
                                                       const real_T re,
                                                       const struct0_T *Temp,
                                                       creal_T k_e[512]);

static __global__ void i_Chi23_Runge_Kuarong_GPU_kerne(const creal_T k_e[512],
                                                       const int32_T iv,
                                                       const int32_T d_dim0,
                                                       emxArray_creal_T d);

static __global__ void
j_Chi23_Runge_Kuarong_GPU_kerne(const emxArray_creal_T d, const int32_T inembed,
                                const real_T ar, const creal_T E_temp[512],
                                const struct0_T *Temp, const int32_T d_dim0,
                                creal_T b_E_temp[512]);

static __global__ void
k_Chi23_Runge_Kuarong_GPU_kerne(const creal_T E_temp[512],
                                const struct0_T *Temp, creal_T b_E_temp[512]);

static __global__ void
l_Chi23_Runge_Kuarong_GPU_kerne(const creal_T exp_minus_omega_contents[1536],
                                const int32_T inembed,
                                const creal_T E_temp[512], creal_T Psio[256]);

static __global__ void m_Chi23_Runge_Kuarong_GPU_kerne(
    const creal_T exp_minus_omega_contents[1536], const int32_T inembed,
    const creal_T E_temp[512], creal_T Psie[256], creal_T Psio[256]);

static __global__ void n_Chi23_Runge_Kuarong_GPU_kerne(creal_T Psie[256]);

static __global__ void o_Chi23_Runge_Kuarong_GPU_kerne(creal_T k_e[512]);

static __global__ void p_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void q_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psie[256],
                                                       const creal_T Psio[256],
                                                       creal_T x[256]);

static __global__ void r_Chi23_Runge_Kuarong_GPU_kerne(
    const creal_T Psio[256], const creal_T x[256], const struct0_T *Temp,
    const creal_T exp_plus_omega_contents[1536], const int32_T inembed,
    creal_T k_e[512], creal_T b_x[256]);

static __global__ void s_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psio[256],
                                                       creal_T Psie[256]);

static __global__ void
t_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psie[256], const creal_T x[256],
                                const struct0_T *Temp,
                                const creal_T exp_plus_omega_contents[1536],
                                const int32_T inembed, creal_T k_e[512]);

static __global__ void u_Chi23_Runge_Kuarong_GPU_kerne(const real_T ar,
                                                       const real_T *q,
                                                       const struct0_T *Temp,
                                                       creal_T k_e[512]);

static __global__ void v_Chi23_Runge_Kuarong_GPU_kerne(const creal_T k_e[512],
                                                       const int32_T inembed,
                                                       const int32_T iv1,
                                                       const int32_T d_dim0,
                                                       emxArray_creal_T d);

static __global__ void w_Chi23_Runge_Kuarong_GPU_kerne(const creal_T F_e[512],
                                                       creal_T E_temp[512]);

static __global__ void
x_Chi23_Runge_Kuarong_GPU_kerne(const emxArray_creal_T d, const int32_T inembed,
                                const real_T ar, const creal_T F_e[512],
                                const struct0_T *Temp, const int32_T d_dim0,
                                creal_T E_temp[512]);

static __global__ void
y_Chi23_Runge_Kuarong_GPU_kerne(const creal_T E_temp[512],
                                const struct0_T *Temp, creal_T F_e[512]);

// Function Definitions
static __global__ __launch_bounds__(
    1024, 1) void Chi23_Runge_Kuarong_GPU_kernel1(const int32_T Runge,
                                                  emxArray_cuint8_T d)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(Runge);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    d.data[k].re = static_cast<uint8_T>(0U);
    d.data[k].im = static_cast<uint8_T>(0U);
  }
}

static __global__
    __launch_bounds__(512, 1) void Chi23_Runge_Kuarong_GPU_kernel2(
        const real_T Par_dt, const struct0_T *Temp, const struct7_T Runge,
        creal_T exp_plus_omega_contents[1536],
        creal_T exp_minus_omega_contents[1536])
{
  uint64_T threadId;
  int32_T i;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId % 512ULL);
  i = static_cast<int32_T>((threadId - static_cast<uint64_T>(k)) / 512ULL);
  if ((static_cast<int32_T>(i < 3)) && (static_cast<int32_T>(k < 512))) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    real_T re;
    d = Temp->Eq.L[k].re;
    d1 = Temp->Eq.L[k].im;
    re = Par_dt * (Runge.a[i] * (0.0 * d - (-d1)));
    im = Par_dt * (Runge.a[i] * (0.0 * d1 + -d));
    if (im == 0.0) {
      re = exp(re);
      im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(im))) &&
                                     (static_cast<int32_T>(isinf(re))))) &&
               (static_cast<int32_T>(re < 0.0))) {
      re = 0.0;
      im = 0.0;
    } else {
      r = exp(re / 2.0);
      re = r * (r * cos(im));
      im = r * (r * sin(im));
    }
    exp_minus_omega_contents[i + 3 * k].re = re;
    exp_minus_omega_contents[i + 3 * k].im = im;
    re = Par_dt * (Runge.a[i] * (0.0 * d - d1));
    im = Par_dt * (Runge.a[i] * (0.0 * d1 + d));
    if (im == 0.0) {
      re = exp(re);
      im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(im))) &&
                                     (static_cast<int32_T>(isinf(re))))) &&
               (static_cast<int32_T>(re < 0.0))) {
      re = 0.0;
      im = 0.0;
    } else {
      r = exp(re / 2.0);
      re = r * (r * cos(im));
      im = r * (r * sin(im));
    }
    exp_plus_omega_contents[i + 3 * k].re = re;
    exp_plus_omega_contents[i + 3 * k].im = im;
  }
}

static __global__
    __launch_bounds__(512, 1) void Chi23_Runge_Kuarong_GPU_kernel3(
        const struct0_T *Temp, creal_T F_e[512], creal_T y[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T im;
    real_T re;
    // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    re = Temp->Par.dt * (0.0 * Temp->Eq.L[k].re - (-Temp->Eq.L[k].im));
    im = Temp->Par.dt * (0.0 * Temp->Eq.L[k].im + -Temp->Eq.L[k].re);
    y[k].re = re;
    y[k].im = im;
    if (im == 0.0) {
      y[k].re = exp(re);
      y[k].im = 0.0;
    } else if ((static_cast<int32_T>((static_cast<int32_T>(isinf(im))) &&
                                     (static_cast<int32_T>(isinf(re))))) &&
               (static_cast<int32_T>(re < 0.0))) {
      y[k].re = 0.0;
      y[k].im = 0.0;
    } else {
      re = exp(re / 2.0);
      y[k].re = re * (re * cos(im));
      y[k].im = re * (re * sin(im));
    }
    F_e[k] = Temp->In.Psi_Start[k];
  }
}

static __global__ __launch_bounds__(32, 1) void Chi23_Runge_Kuarong_GPU_kernel4(
    const struct0_T *Temp, int32_T *i)
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    *i = static_cast<int32_T>(Temp->Par.T / Temp->Par.dt);
  }
}

static __global__
    __launch_bounds__(1024, 1) void Chi23_Runge_Kuarong_GPU_kernel5(
        const emxArray_cuint8_T d, const int32_T b_d, emxArray_creal_T c_d)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(b_d);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    c_d.data[k].re = 0.0;
    c_d.data[k].im = static_cast<real_T>(d.data[k].im);
  }
}

static __global__ __launch_bounds__(
    512, 1) void Chi23_Runge_Kuarong_GPU_kernel6(const creal_T F_e[512],
                                                 creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    E_temp[k] = F_e[k];
  }
}

static __global__
    __launch_bounds__(256, 1) void Chi23_Runge_Kuarong_GPU_kernel7(
        const creal_T exp_minus_omega_contents[1536], const creal_T F_e[512],
        creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    d = F_e[k].re;
    d1 = F_e[k].im;
    im = exp_minus_omega_contents[3 * k].re;
    r = exp_minus_omega_contents[3 * k].im;
    Psio[k].re = d * im - d1 * r;
    Psio[k].im = d * r + d1 * im;
  }
}

static __global__
    __launch_bounds__(256, 1) void Chi23_Runge_Kuarong_GPU_kernel8(
        const creal_T exp_minus_omega_contents[1536], const creal_T F_e[512],
        creal_T Psie[256], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    r = Psio[k].re;
    im = Psio[k].im;
    if (im == 0.0) {
      r /= 256.0;
      im = 0.0;
    } else if (r == 0.0) {
      r = 0.0;
      im /= 256.0;
    } else {
      r /= 256.0;
      im /= 256.0;
    }
    Psio[k].re = r;
    Psio[k].im = im;
    d = F_e[k + 256].re;
    d1 = F_e[k + 256].im;
    im = exp_minus_omega_contents[3 * (k + 256)].re;
    r = exp_minus_omega_contents[3 * (k + 256)].im;
    Psie[k].re = d * im - d1 * r;
    Psie[k].im = d * r + d1 * im;
  }
}

static __global__
    __launch_bounds__(256,
                      1) void Chi23_Runge_Kuarong_GPU_kernel9(creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T im;
    real_T r;
    r = Psie[k].re;
    im = Psie[k].im;
    if (im == 0.0) {
      r /= 256.0;
      im = 0.0;
    } else if (r == 0.0) {
      r = 0.0;
      im /= 256.0;
    } else {
      r /= 256.0;
      im /= 256.0;
    }
    Psie[k].re = r;
    Psie[k].im = im;
  }
}

static __global__ __launch_bounds__(
    512, 1) void ab_Chi23_Runge_Kuarong_GPU_kern(const creal_T F_e[512],
                                                 const creal_T y[512],
                                                 const struct0_T *Temp,
                                                 creal_T b_y[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T d;
    d = Temp->Eq.mode_range[k];
    b_y[k].re =
        y[static_cast<int32_T>(d) - 1].re *
            F_e[static_cast<int32_T>(d) - 1].re -
        y[static_cast<int32_T>(d) - 1].im * F_e[static_cast<int32_T>(d) - 1].im;
    b_y[k].im =
        y[static_cast<int32_T>(d) - 1].re *
            F_e[static_cast<int32_T>(d) - 1].im +
        y[static_cast<int32_T>(d) - 1].im * F_e[static_cast<int32_T>(d) - 1].re;
  }
}

static __global__
    __launch_bounds__(512,
                      1) void b_Chi23_Runge_Kuarong_GPU_kerne(creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    k_e[k].re = 0.0;
    k_e[k].im = 0.0;
  }
}

static __global__
    __launch_bounds__(512, 1) void bb_Chi23_Runge_Kuarong_GPU_kern(
        const creal_T y[512], const struct0_T *Temp, creal_T F_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    F_e[static_cast<int32_T>(Temp->Eq.mode_range[k]) - 1] = y[k];
  }
}

static __global__
    __launch_bounds__(256, 1) void c_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    im = Psio[k].re;
    r = -Psio[k].im;
    d = Psie[k].re;
    d1 = Psie[k].im;
    x[k].re = im * d - r * d1;
    x[k].im = im * d1 + r * d;
  }
}

static __global__
    __launch_bounds__(256, 1) void d_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    d = Psio[k].re;
    d1 = Psio[k].im;
    im = hypot(d, d1);
    r = im * im;
    im = hypot(Psie[k].re, Psie[k].im);
    x[k].re = (r + 2.0 * (im * im)) * d;
    x[k].im = (r + 2.0 * (im * im)) * d1;
  }
}

static __global__
    __launch_bounds__(256, 1) void e_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psio[256], const creal_T x[256], const struct0_T *Temp,
        const creal_T exp_plus_omega_contents[1536], creal_T k_e[512],
        creal_T b_x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T Temp_re;
    real_T d;
    real_T d1;
    real_T im;
    real_T re;
    //
    d = exp_plus_omega_contents[3 * k].re;
    d1 = exp_plus_omega_contents[3 * k].im;
    im = 0.0 * d - d1;
    re = 0.0 * d1 + d;
    d = Temp->Eq.gam2o[k];
    d1 = Temp->Eq.gam3o[k];
    Temp_re = d * b_x[k].re + d1 * x[k].re;
    d = d * b_x[k].im + d1 * x[k].im;
    k_e[k].re = im * Temp_re - re * d;
    k_e[k].im = im * d + re * Temp_re;
    d = Psio[k].re;
    d1 = Psio[k].im;
    b_x[k].re = d * d - d1 * d1;
    b_x[k].im = d * d1 + d1 * d;
  }
}

static __global__ __launch_bounds__(
    256, 1) void f_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psio[256],
                                                 creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    d = hypot(Psio[k].re, Psio[k].im);
    d1 = d * d;
    im = Psie[k].re;
    r = Psie[k].im;
    d = hypot(im, r);
    im *= 2.0 * d1 + d * d;
    Psie[k].re = im;
    r *= 2.0 * d1 + d * d;
    Psie[k].im = r;
  }
}

static __global__
    __launch_bounds__(256, 1) void g_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T x[256], const struct0_T *Temp,
        const creal_T exp_plus_omega_contents[1536], creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T Temp_re;
    real_T d;
    real_T d1;
    real_T im;
    real_T re;
    d = exp_plus_omega_contents[3 * (k + 256)].re;
    d1 = exp_plus_omega_contents[3 * (k + 256)].im;
    im = 0.0 * d - d1;
    re = 0.0 * d1 + d;
    d = Temp->Eq.gam2e[k];
    d1 = Temp->Eq.gam3e[k];
    Temp_re = d * x[k].re + d1 * Psie[k].re;
    d = d * x[k].im + d1 * Psie[k].im;
    k_e[k + 256].re = im * Temp_re - re * d;
    k_e[k + 256].im = im * d + re * Temp_re;
  }
}

static void gpuEmxEnsureCapacity_creal_T(const emxArray_creal_T *cpu,
                                         emxArray_creal_T *gpu)
{
  creal_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(creal_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(creal_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(creal_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxEnsureCapacity_cuint8_T(const emxArray_cuint8_T *cpu,
                                          emxArray_cuint8_T *gpu)
{
  cuint8_T *newData;
  if (gpu->data == 0) {
    newData = 0ULL;
    hipMalloc(&newData, cpu->allocatedSize * sizeof(cuint8_T));
    gpu->numDimensions = cpu->numDimensions;
    gpu->size = (int32_T *)emlrtCallocMex(
        static_cast<uint32_T>(gpu->numDimensions), sizeof(int32_T));
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      gpu->size[i] = cpu->size[i];
    }
    gpu->allocatedSize = cpu->allocatedSize;
    gpu->canFreeData = true;
    gpu->data = newData;
  } else {
    int32_T actualSizeCpu;
    int32_T actualSizeGpu;
    actualSizeCpu = 1;
    actualSizeGpu = 1;
    for (int32_T i{0}; i < cpu->numDimensions; i++) {
      actualSizeGpu *= gpu->size[i];
      actualSizeCpu *= cpu->size[i];
      gpu->size[i] = cpu->size[i];
    }
    if (gpu->allocatedSize < actualSizeCpu) {
      newData = 0ULL;
      hipMalloc(&newData, cpu->allocatedSize * sizeof(cuint8_T));
      hipMemcpy(newData, gpu->data, actualSizeGpu * sizeof(cuint8_T),
                 hipMemcpyDeviceToDevice);
      gpu->allocatedSize = cpu->allocatedSize;
      if (gpu->canFreeData) {
        hipFree(gpu->data);
      }
      gpu->canFreeData = true;
      gpu->data = newData;
    }
  }
}

static void gpuEmxFree_creal_T(emxArray_creal_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxFree_cuint8_T(emxArray_cuint8_T *gpu)
{
  if (gpu->data != (void *)4207599121ULL) {
    hipFree(gpu->data);
  }
  emlrtFreeMex(gpu->size);
}

static void gpuEmxReset_creal_T(emxArray_creal_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_creal_T));
}

static void gpuEmxReset_cuint8_T(emxArray_cuint8_T *gpu)
{
  std::memset(gpu, 0, sizeof(emxArray_cuint8_T));
}

static __global__ __launch_bounds__(32, 1) void h_Chi23_Runge_Kuarong_GPU_kerne(
    const real_T im, const real_T re, const struct0_T *Temp, creal_T k_e[512])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    //
    k_e[0].re += Temp->Eq.ko * re * Temp->Eq.H_f;
    k_e[0].im += Temp->Eq.ko * im * Temp->Eq.H_f;
  }
}

static __global__ __launch_bounds__(
    1024, 1) void i_Chi23_Runge_Kuarong_GPU_kerne(const creal_T k_e[512],
                                                  const int32_T iv,
                                                  const int32_T d_dim0,
                                                  emxArray_creal_T d)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(iv);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    d.data[d_dim0 * k] = k_e[k];
  }
}

static __global__
    __launch_bounds__(512, 1) void j_Chi23_Runge_Kuarong_GPU_kerne(
        const emxArray_creal_T d, const int32_T inembed, const real_T ar,
        const creal_T E_temp[512], const struct0_T *Temp, const int32_T d_dim0,
        creal_T b_E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T b_d;
    b_d = Temp->Eq.mode_range[k];
    b_E_temp[k].re =
        E_temp[static_cast<int32_T>(b_d) - 1].re +
        ar * d.data[inembed + d_dim0 * (static_cast<int32_T>(b_d) - 1)].re;
    b_E_temp[k].im =
        E_temp[static_cast<int32_T>(b_d) - 1].im +
        ar * d.data[inembed + d_dim0 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__
    __launch_bounds__(512, 1) void k_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T E_temp[512], const struct0_T *Temp, creal_T b_E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    b_E_temp[static_cast<int32_T>(Temp->Eq.mode_range[k]) - 1] = E_temp[k];
  }
}

static __global__
    __launch_bounds__(256, 1) void l_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T exp_minus_omega_contents[1536], const int32_T inembed,
        const creal_T E_temp[512], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    d = E_temp[k].re;
    d1 = E_temp[k].im;
    Psio[k].re = d * exp_minus_omega_contents[inembed + 3 * k].re -
                 d1 * exp_minus_omega_contents[inembed + 3 * k].im;
    Psio[k].im = d * exp_minus_omega_contents[inembed + 3 * k].im +
                 d1 * exp_minus_omega_contents[inembed + 3 * k].re;
  }
}

static __global__
    __launch_bounds__(256, 1) void m_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T exp_minus_omega_contents[1536], const int32_T inembed,
        const creal_T E_temp[512], creal_T Psie[256], creal_T Psio[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T im;
    real_T r;
    r = Psio[k].re;
    im = Psio[k].im;
    if (im == 0.0) {
      r /= 256.0;
      im = 0.0;
    } else if (r == 0.0) {
      r = 0.0;
      im /= 256.0;
    } else {
      r /= 256.0;
      im /= 256.0;
    }
    Psio[k].re = r;
    Psio[k].im = im;
    r = E_temp[k + 256].re;
    im = E_temp[k + 256].im;
    Psie[k].re = r * exp_minus_omega_contents[inembed + 3 * (k + 256)].re -
                 im * exp_minus_omega_contents[inembed + 3 * (k + 256)].im;
    Psie[k].im = r * exp_minus_omega_contents[inembed + 3 * (k + 256)].im +
                 im * exp_minus_omega_contents[inembed + 3 * (k + 256)].re;
  }
}

static __global__
    __launch_bounds__(256,
                      1) void n_Chi23_Runge_Kuarong_GPU_kerne(creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T im;
    real_T r;
    r = Psie[k].re;
    im = Psie[k].im;
    if (im == 0.0) {
      r /= 256.0;
      im = 0.0;
    } else if (r == 0.0) {
      r = 0.0;
      im /= 256.0;
    } else {
      r /= 256.0;
      im /= 256.0;
    }
    Psie[k].re = r;
    Psie[k].im = im;
  }
}

static __global__
    __launch_bounds__(512,
                      1) void o_Chi23_Runge_Kuarong_GPU_kerne(creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    k_e[k].re = 0.0;
    k_e[k].im = 0.0;
  }
}

static __global__
    __launch_bounds__(256, 1) void p_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    im = Psio[k].re;
    r = -Psio[k].im;
    d = Psie[k].re;
    d1 = Psie[k].im;
    x[k].re = im * d - r * d1;
    x[k].im = im * d1 + r * d;
  }
}

static __global__
    __launch_bounds__(256, 1) void q_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T Psio[256], creal_T x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    d = Psio[k].re;
    d1 = Psio[k].im;
    im = hypot(d, d1);
    r = im * im;
    im = hypot(Psie[k].re, Psie[k].im);
    x[k].re = (r + 2.0 * (im * im)) * d;
    x[k].im = (r + 2.0 * (im * im)) * d1;
  }
}

static __global__
    __launch_bounds__(256, 1) void r_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psio[256], const creal_T x[256], const struct0_T *Temp,
        const creal_T exp_plus_omega_contents[1536], const int32_T inembed,
        creal_T k_e[512], creal_T b_x[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T Temp_re;
    real_T d;
    real_T d1;
    real_T im;
    real_T re;
    //
    im = 0.0 * exp_plus_omega_contents[inembed + 3 * k].re -
         exp_plus_omega_contents[inembed + 3 * k].im;
    re = 0.0 * exp_plus_omega_contents[inembed + 3 * k].im +
         exp_plus_omega_contents[inembed + 3 * k].re;
    d = Temp->Eq.gam2o[k];
    d1 = Temp->Eq.gam3o[k];
    Temp_re = d * b_x[k].re + d1 * x[k].re;
    d = d * b_x[k].im + d1 * x[k].im;
    k_e[k].re = im * Temp_re - re * d;
    k_e[k].im = im * d + re * Temp_re;
    d = Psio[k].re;
    d1 = Psio[k].im;
    b_x[k].re = d * d - d1 * d1;
    b_x[k].im = d * d1 + d1 * d;
  }
}

static __global__ __launch_bounds__(
    256, 1) void s_Chi23_Runge_Kuarong_GPU_kerne(const creal_T Psio[256],
                                                 creal_T Psie[256])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T d;
    real_T d1;
    real_T im;
    real_T r;
    d = hypot(Psio[k].re, Psio[k].im);
    d1 = d * d;
    im = Psie[k].re;
    r = Psie[k].im;
    d = hypot(im, r);
    im *= 2.0 * d1 + d * d;
    Psie[k].re = im;
    r *= 2.0 * d1 + d * d;
    Psie[k].im = r;
  }
}

static __global__
    __launch_bounds__(256, 1) void t_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T Psie[256], const creal_T x[256], const struct0_T *Temp,
        const creal_T exp_plus_omega_contents[1536], const int32_T inembed,
        creal_T k_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 256) {
    real_T Temp_re;
    real_T d;
    real_T d1;
    real_T im;
    real_T re;
    im = 0.0 * exp_plus_omega_contents[inembed + 3 * (k + 256)].re -
         exp_plus_omega_contents[inembed + 3 * (k + 256)].im;
    re = 0.0 * exp_plus_omega_contents[inembed + 3 * (k + 256)].im +
         exp_plus_omega_contents[inembed + 3 * (k + 256)].re;
    d = Temp->Eq.gam2e[k];
    d1 = Temp->Eq.gam3e[k];
    Temp_re = d * x[k].re + d1 * Psie[k].re;
    d = d * x[k].im + d1 * Psie[k].im;
    k_e[k + 256].re = im * Temp_re - re * d;
    k_e[k + 256].im = im * d + re * Temp_re;
  }
}

static __global__ __launch_bounds__(32, 1) void u_Chi23_Runge_Kuarong_GPU_kerne(
    const real_T ar, const real_T *q, const struct0_T *Temp, creal_T k_e[512])
{
  uint64_T threadId;
  int32_T tmpIdx;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  tmpIdx = static_cast<int32_T>(threadId);
  if (tmpIdx < 1) {
    k_e[0].re += Temp->Eq.ko * *q * Temp->Eq.H_f;
    k_e[0].im += Temp->Eq.ko * ar * Temp->Eq.H_f;
  }
}

static __global__
    __launch_bounds__(1024, 1) void v_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T k_e[512], const int32_T inembed, const int32_T iv1,
        const int32_T d_dim0, emxArray_creal_T d)
{
  uint64_T loopEnd;
  uint64_T threadId;
  uint64_T threadStride;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  threadStride = mwGetTotalThreadsLaunched();
  loopEnd = static_cast<uint64_T>(iv1);
  for (uint64_T idx{threadId}; idx <= loopEnd; idx += threadStride) {
    int32_T k;
    k = static_cast<int32_T>(idx);
    d.data[inembed + d_dim0 * k] = k_e[k];
  }
}

static __global__ __launch_bounds__(
    512, 1) void w_Chi23_Runge_Kuarong_GPU_kerne(const creal_T F_e[512],
                                                 creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    E_temp[k] = F_e[k];
  }
}

static __global__
    __launch_bounds__(512, 1) void x_Chi23_Runge_Kuarong_GPU_kerne(
        const emxArray_creal_T d, const int32_T inembed, const real_T ar,
        const creal_T F_e[512], const struct0_T *Temp, const int32_T d_dim0,
        creal_T E_temp[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    real_T b_d;
    b_d = Temp->Eq.mode_range[k];
    E_temp[k].re =
        F_e[static_cast<int32_T>(b_d) - 1].re +
        ar * d.data[inembed + d_dim0 * (static_cast<int32_T>(b_d) - 1)].re;
    E_temp[k].im =
        F_e[static_cast<int32_T>(b_d) - 1].im +
        ar * d.data[inembed + d_dim0 * (static_cast<int32_T>(b_d) - 1)].im;
  }
}

static __global__
    __launch_bounds__(512, 1) void y_Chi23_Runge_Kuarong_GPU_kerne(
        const creal_T E_temp[512], const struct0_T *Temp, creal_T F_e[512])
{
  uint64_T threadId;
  int32_T k;
  threadId = static_cast<uint64_T>(mwGetGlobalThreadIndexInXDimension());
  k = static_cast<int32_T>(threadId);
  if (k < 512) {
    F_e[static_cast<int32_T>(Temp->Eq.mode_range[k]) - 1] = E_temp[k];
  }
}

void Chi23_Runge_Kuarong_GPU(const struct0_T *Temp, const struct7_T *Runge,
                             struct8_T *Sol)
{
  hipfftType b_fftType;
  hipfftType c_fftType;
  hipfftType d_fftType;
  hipfftType e_fftType;
  hipfftType f_fftType;
  hipfftType fftType;
  hipfftType g_fftType;
  hipfftType h_fftType;
  hipfftType i_fftType;
  hipfftType j_fftType;
  hipfftType k_fftType;
  hipfftType l_fftType;
  int b_fftDirection;
  int c_fftDirection;
  int d_fftDirection;
  int e_fftDirection;
  int f_fftDirection;
  int fftDirection;
  int g_fftDirection;
  int h_fftDirection;
  int i_fftDirection;
  int j_fftDirection;
  int k_fftDirection;
  int l_fftDirection;
  dim3 block;
  dim3 grid;
  emxArray_creal_T b_gpu_d;
  emxArray_creal_T *b_d;
  emxArray_cuint8_T gpu_d;
  emxArray_cuint8_T *d;
  struct0_T *gpu_Temp;
  creal_T exp_plus_omega_contents[1536];
  creal_T(*gpu_exp_minus_omega_contents)[1536];
  creal_T(*gpu_exp_plus_omega_contents)[1536];
  creal_T F_e[512];
  creal_T(*b_gpu_E_temp)[512];
  creal_T(*b_gpu_y)[512];
  creal_T(*gpu_E_temp)[512];
  creal_T(*gpu_F_e)[512];
  creal_T(*gpu_k_e)[512];
  creal_T(*gpu_y)[512];
  creal_T(*b_gpu_x)[256];
  creal_T(*gpu_Psie)[256];
  creal_T(*gpu_Psio)[256];
  creal_T(*gpu_x)[256];
  real_T Par_dt;
  real_T ai;
  real_T ar;
  real_T im;
  real_T q;
  real_T re;
  real_T y;
  real_T *gpu_q;
  int32_T iv[2];
  int32_T iv1[2];
  int32_T b_i;
  int32_T i;
  int32_T i1;
  int32_T i2;
  int32_T i3;
  int32_T *gpu_i;
  boolean_T c_exp_plus_omega_contents_dirty;
  boolean_T validLaunchParams;
  hipMalloc(&gpu_q, 8ULL);
  hipMalloc(&b_gpu_E_temp, 8192ULL);
  hipMalloc(&b_gpu_y, 8192ULL);
  hipMalloc(&b_gpu_x, 4096ULL);
  hipMalloc(&gpu_x, 4096ULL);
  hipMalloc(&gpu_k_e, 8192ULL);
  hipMalloc(&gpu_Psie, 4096ULL);
  hipMalloc(&gpu_Psio, 4096ULL);
  hipMalloc(&gpu_E_temp, 8192ULL);
  gpuEmxReset_creal_T(&b_gpu_d);
  hipMalloc(&gpu_i, 4ULL);
  hipMalloc(&gpu_y, 8192ULL);
  hipMalloc(&gpu_F_e, 8192ULL);
  hipMalloc(&gpu_exp_minus_omega_contents, 24576ULL);
  hipMalloc(&gpu_exp_plus_omega_contents, 24576ULL);
  hipMalloc(&gpu_Temp, 114232ULL);
  gpuEmxReset_cuint8_T(&gpu_d);
  emlrtHeapReferenceStackEnterFcnR2012b(emlrtRootTLSGlobal);
  emxInit_cuint8_T(&d, 2, true);
  //     coder.gpu.iterations
  i = d->size[0] * d->size[1];
  d->size[0] = static_cast<int32_T>(Runge->s);
  d->size[1] = static_cast<int32_T>(2.0 * Temp->Space.N);
  emxEnsureCapacity_cuint8_T(d, i);
  gpuEmxEnsureCapacity_cuint8_T(d, &gpu_d);
  i = static_cast<int32_T>(Runge->s) *
          static_cast<int32_T>(2.0 * Temp->Space.N) -
      1;
  validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i + 1LL),
                                              &grid, &block, 1024U, 65535U);
  if (validLaunchParams) {
    Chi23_Runge_Kuarong_GPU_kernel1<<<grid, block>>>(i, gpu_d);
  }
  Par_dt = Temp->Par.dt;
  // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
  hipMemcpy(gpu_Temp, Temp, 114232ULL, hipMemcpyHostToDevice);
  Chi23_Runge_Kuarong_GPU_kernel2<<<dim3(3U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      Par_dt, gpu_Temp, *Runge, *gpu_exp_plus_omega_contents,
      *gpu_exp_minus_omega_contents);
  c_exp_plus_omega_contents_dirty = true;
  // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
  Chi23_Runge_Kuarong_GPU_kernel3<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
      gpu_Temp, *gpu_F_e, *gpu_y);
  i = Sol->Psio->size[0] * Sol->Psio->size[1];
  Sol->Psio->size[0] = static_cast<int32_T>(Temp->Par.dd);
  Sol->Psio->size[1] = static_cast<int32_T>(Temp->Space.N);
  emxEnsureCapacity_creal_T(Sol->Psio, i);
  i = Sol->Psie->size[0] * Sol->Psie->size[1];
  Sol->Psie->size[0] = static_cast<int32_T>(Temp->Par.dd);
  Sol->Psie->size[1] = static_cast<int32_T>(Temp->Space.N);
  emxEnsureCapacity_creal_T(Sol->Psie, i);
  for (i = 0; i < static_cast<int32_T>(Temp->Par.dd) *
                      static_cast<int32_T>(Temp->Space.N);
       i++) {
    Sol->Psio->data[i].re = 0.0;
    Sol->Psio->data[i].im = 0.0;
    Sol->Psie->data[i].re = 0.0;
    Sol->Psie->data[i].im = 0.0;
  }
  i = Sol->t->size[0] * Sol->t->size[1];
  Sol->t->size[0] = 1;
  Sol->t->size[1] = static_cast<int32_T>(Temp->Par.dd);
  emxEnsureCapacity_creal_T(Sol->t, i);
  for (i = 0; i < static_cast<int32_T>(Temp->Par.dd); i++) {
    Sol->t->data[i].re = 0.0;
    Sol->t->data[i].im = 0.0;
  }
  Chi23_Runge_Kuarong_GPU_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
      gpu_Temp, gpu_i);
  hipMemcpy(&b_i, gpu_i, 4ULL, hipMemcpyDeviceToHost);
  if (0 <= b_i - 1) {
    fftDirection = HIPFFT_BACKWARD;
    fftType = HIPFFT_Z2Z;
    b_fftDirection = HIPFFT_BACKWARD;
    b_fftType = HIPFFT_Z2Z;
    c_fftDirection = HIPFFT_FORWARD;
    c_fftType = HIPFFT_Z2Z;
    d_fftDirection = HIPFFT_FORWARD;
    d_fftType = HIPFFT_Z2Z;
    e_fftDirection = HIPFFT_FORWARD;
    e_fftType = HIPFFT_Z2Z;
    f_fftDirection = HIPFFT_FORWARD;
    f_fftType = HIPFFT_Z2Z;
    iv[1] = d->size[1];
    i1 = static_cast<int32_T>(Runge->s + -1.0);
    i2 = static_cast<int32_T>(Runge->s);
    y = Temp->Par.s_t / Temp->Par.dt;
  }
  emxInit_creal_T(&b_d, 2, true);
  if (0 <= b_i - 1) {
    hipMemcpy(exp_plus_omega_contents, *gpu_exp_plus_omega_contents, 24576ULL,
               hipMemcpyDeviceToHost);
    ar = 256.0 * exp_plus_omega_contents[0].re;
    c_exp_plus_omega_contents_dirty = false;
    ai = 256.0 * exp_plus_omega_contents[0].im;
    if (ai == 0.0) {
      re = ar / 2.0;
      im = 0.0;
    } else if (ar == 0.0) {
      re = 0.0;
      im = ai / 2.0;
    } else {
      re = ar / 2.0;
      im = ai / 2.0;
    }
  }
  for (int32_T ni{0}; ni < b_i; ni++) {
    hipfftHandle b_fftPlanHandle;
    hipfftHandle c_fftPlanHandle;
    hipfftHandle d_fftPlanHandle;
    hipfftHandle e_fftPlanHandle;
    hipfftHandle f_fftPlanHandle;
    hipfftHandle fftPlanHandle;
    int32_T b_i2;
    int32_T inembed;
    i = b_d->size[0] * b_d->size[1];
    b_d->size[0] = d->size[0];
    b_d->size[1] = d->size[1];
    emxEnsureCapacity_creal_T(b_d, i);
    gpuEmxEnsureCapacity_creal_T(b_d, &b_gpu_d);
    i = d->size[0] * d->size[1] - 1;
    validLaunchParams = mwGetLaunchParameters1D(static_cast<real_T>(i + 1LL),
                                                &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      Chi23_Runge_Kuarong_GPU_kernel5<<<grid, block>>>(gpu_d, i, b_gpu_d);
    }
    // %%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%%
    Chi23_Runge_Kuarong_GPU_kernel6<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_F_e, *gpu_E_temp);
    Chi23_Runge_Kuarong_GPU_kernel7<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_exp_minus_omega_contents, *gpu_F_e, *gpu_Psio);
    inembed = 256;
    fftPlanHandle = acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, fftType, 1);
    hipfftExecZ2Z(fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psio)[0],
                 (hipfftDoubleComplex *)&(*gpu_Psio)[0], fftDirection);
    Chi23_Runge_Kuarong_GPU_kernel8<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_exp_minus_omega_contents, *gpu_F_e, *gpu_Psie, *gpu_Psio);
    inembed = 256;
    b_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, b_fftType, 1);
    hipfftExecZ2Z(b_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                 (hipfftDoubleComplex *)&(*gpu_Psie)[0], b_fftDirection);
    Chi23_Runge_Kuarong_GPU_kernel9<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie);
    b_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_k_e);
    c_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_Psio, *gpu_x);
    inembed = 256;
    c_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, c_fftType, 1);
    hipfftExecZ2Z(c_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                 (hipfftDoubleComplex *)&(*gpu_x)[0], c_fftDirection);
    d_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_Psio, *b_gpu_x);
    inembed = 256;
    d_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, d_fftType, 1);
    hipfftExecZ2Z(d_fftPlanHandle, (hipfftDoubleComplex *)&(*b_gpu_x)[0],
                 (hipfftDoubleComplex *)&(*b_gpu_x)[0], d_fftDirection);
    //
    e_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psio, *b_gpu_x, gpu_Temp, *gpu_exp_plus_omega_contents, *gpu_k_e,
        *gpu_x);
    inembed = 256;
    e_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, e_fftType, 1);
    hipfftExecZ2Z(e_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                 (hipfftDoubleComplex *)&(*gpu_x)[0], e_fftDirection);
    f_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psio, *gpu_Psie);
    inembed = 256;
    f_fftPlanHandle =
        acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, f_fftType, 1);
    hipfftExecZ2Z(f_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                 (hipfftDoubleComplex *)&(*gpu_Psie)[0], f_fftDirection);
    g_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
        *gpu_Psie, *gpu_x, gpu_Temp, *gpu_exp_plus_omega_contents, *gpu_k_e);
    //
    h_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
        im, re, gpu_Temp, *gpu_k_e);
    i = b_d->size[0];
    validLaunchParams = mwGetLaunchParameters1D(
        static_cast<real_T>((iv[1] - 1) + 1LL), &grid, &block, 1024U, 65535U);
    if (validLaunchParams) {
      i_Chi23_Runge_Kuarong_GPU_kerne<<<grid, block>>>(*gpu_k_e, iv[1] - 1, i,
                                                       b_gpu_d);
    }
    if (0 <= i1 - 1) {
      i3 = static_cast<int32_T>(Runge->s - 1.0);
      g_fftDirection = HIPFFT_BACKWARD;
      g_fftType = HIPFFT_Z2Z;
      h_fftDirection = HIPFFT_BACKWARD;
      h_fftType = HIPFFT_Z2Z;
      i_fftDirection = HIPFFT_FORWARD;
      i_fftType = HIPFFT_Z2Z;
      j_fftDirection = HIPFFT_FORWARD;
      j_fftType = HIPFFT_Z2Z;
      k_fftDirection = HIPFFT_FORWARD;
      k_fftType = HIPFFT_Z2Z;
      l_fftDirection = HIPFFT_FORWARD;
      l_fftType = HIPFFT_Z2Z;
    }
    for (b_i2 = 0; b_i2 < i1; b_i2++) {
      for (inembed = 0; inembed < i3; inembed++) {
        ar = Runge->b[(b_i2 + 3 * inembed) + 1];
        if (ar != 0.0) {
          ar *= Par_dt;
          i = b_d->size[0];
          j_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              b_gpu_d, inembed, ar, *gpu_E_temp, gpu_Temp, i, *b_gpu_E_temp);
          k_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U),
                                            dim3(512U, 1U, 1U)>>>(
              *b_gpu_E_temp, gpu_Temp, *gpu_E_temp);
        }
      }
      hipfftHandle g_fftPlanHandle;
      hipfftHandle h_fftPlanHandle;
      hipfftHandle i_fftPlanHandle;
      hipfftHandle j_fftPlanHandle;
      hipfftHandle k_fftPlanHandle;
      hipfftHandle l_fftPlanHandle;
      l_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_exp_minus_omega_contents, b_i2 + 1, *gpu_E_temp, *gpu_Psio);
      inembed = 256;
      g_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, g_fftType, 1);
      hipfftExecZ2Z(g_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psio)[0],
                   (hipfftDoubleComplex *)&(*gpu_Psio)[0], g_fftDirection);
      m_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_exp_minus_omega_contents, b_i2 + 1, *gpu_E_temp, *gpu_Psie,
          *gpu_Psio);
      inembed = 256;
      h_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, h_fftType, 1);
      hipfftExecZ2Z(h_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                   (hipfftDoubleComplex *)&(*gpu_Psie)[0], h_fftDirection);
      n_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie);
      o_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          *gpu_k_e);
      p_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_Psio, *gpu_x);
      inembed = 256;
      i_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, i_fftType, 1);
      hipfftExecZ2Z(i_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                   (hipfftDoubleComplex *)&(*gpu_x)[0], i_fftDirection);
      q_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_Psio, *b_gpu_x);
      inembed = 256;
      j_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, j_fftType, 1);
      hipfftExecZ2Z(j_fftPlanHandle, (hipfftDoubleComplex *)&(*b_gpu_x)[0],
                   (hipfftDoubleComplex *)&(*b_gpu_x)[0], j_fftDirection);
      //
      r_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psio, *b_gpu_x, gpu_Temp, *gpu_exp_plus_omega_contents, b_i2 + 1,
          *gpu_k_e, *gpu_x);
      inembed = 256;
      k_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, k_fftType, 1);
      hipfftExecZ2Z(k_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_x)[0],
                   (hipfftDoubleComplex *)&(*gpu_x)[0], k_fftDirection);
      s_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psio, *gpu_Psie);
      inembed = 256;
      l_fftPlanHandle =
          acquireCUFFTPlan(1, &inembed, &inembed, 1, 1, l_fftType, 1);
      hipfftExecZ2Z(l_fftPlanHandle, (hipfftDoubleComplex *)&(*gpu_Psie)[0],
                   (hipfftDoubleComplex *)&(*gpu_Psie)[0], l_fftDirection);
      t_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(256U, 1U, 1U)>>>(
          *gpu_Psie, *gpu_x, gpu_Temp, *gpu_exp_plus_omega_contents, b_i2 + 1,
          *gpu_k_e);
      //
      if (c_exp_plus_omega_contents_dirty) {
        hipMemcpy(exp_plus_omega_contents, *gpu_exp_plus_omega_contents,
                   24576ULL, hipMemcpyDeviceToHost);
      }
      ar = 256.0 * exp_plus_omega_contents[b_i2 + 1].re;
      c_exp_plus_omega_contents_dirty = false;
      ai = 256.0 * exp_plus_omega_contents[b_i2 + 1].im;
      if (ai == 0.0) {
        q = ar / 2.0;
        ar = 0.0;
      } else if (ar == 0.0) {
        q = 0.0;
        ar = ai / 2.0;
      } else {
        q = ar / 2.0;
        ar = ai / 2.0;
      }
      hipMemcpy(gpu_q, &q, 8ULL, hipMemcpyHostToDevice);
      u_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(
          ar, gpu_q, gpu_Temp, *gpu_k_e);
      iv1[1] = b_d->size[1];
      i = b_d->size[0];
      validLaunchParams =
          mwGetLaunchParameters1D(static_cast<real_T>((iv1[1] - 1) + 1LL),
                                  &grid, &block, 1024U, 65535U);
      if (validLaunchParams) {
        v_Chi23_Runge_Kuarong_GPU_kerne<<<grid, block>>>(
            *gpu_k_e, b_i2 + 1, iv1[1] - 1, i, b_gpu_d);
      }
      w_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
          *gpu_F_e, *gpu_E_temp);
    }
    for (inembed = 0; inembed < i2; inembed++) {
      ar = Runge->c[inembed];
      if (ar != 0.0) {
        ar *= Par_dt;
        i = b_d->size[0];
        x_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            b_gpu_d, inembed, ar, *gpu_F_e, gpu_Temp, i, *b_gpu_E_temp);
        y_Chi23_Runge_Kuarong_GPU_kerne<<<dim3(1U, 1U, 1U),
                                          dim3(512U, 1U, 1U)>>>(
            *b_gpu_E_temp, gpu_Temp, *gpu_F_e);
      }
    }
    ab_Chi23_Runge_Kuarong_GPU_kern<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *gpu_F_e, *gpu_y, gpu_Temp, *b_gpu_y);
    bb_Chi23_Runge_Kuarong_GPU_kern<<<dim3(1U, 1U, 1U), dim3(512U, 1U, 1U)>>>(
        *b_gpu_y, gpu_Temp, *gpu_F_e);
    validLaunchParams = true;
    // ,Runge,exp_plus_omega,exp_minus_omega,Temp,shift_back
    ar = static_cast<real_T>(ni) + 1.0;
    if (!(y == 0.0)) {
      if (std::isnan(y)) {
        ar = rtNaN;
      } else if (std::isinf(y)) {
        if (y < 0.0) {
          ar = y;
        }
      } else {
        boolean_T rEQ0;
        ar = std::fmod(static_cast<real_T>(ni) + 1.0, y);
        rEQ0 = (ar == 0.0);
        if ((!rEQ0) && (y > std::floor(y))) {
          q = std::abs((static_cast<real_T>(ni) + 1.0) / y);
          rEQ0 =
              !(std::abs(q - std::floor(q + 0.5)) > 2.2204460492503131E-16 * q);
        }
        if (rEQ0) {
          ar = y * 0.0;
        } else if (y < 0.0) {
          ar += y;
        }
      }
    }
    if (ar == 0.0) {
      q = std::round((static_cast<real_T>(ni) + 1.0) * Temp->Par.dt /
                     Temp->Par.s_t);
      if (1.0 > Temp->Space.N) {
        inembed = 0;
      } else {
        inembed = static_cast<int32_T>(Temp->Space.N);
      }
      for (i = 0; i < inembed; i++) {
        if (validLaunchParams) {
          hipMemcpy(F_e, *gpu_F_e, 8192ULL, hipMemcpyDeviceToHost);
        }
        ar = F_e[i].re;
        validLaunchParams = false;
        ai = F_e[i].im;
        if (ai == 0.0) {
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .re = ar / Temp->Space.N;
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .im = 0.0;
        } else if (ar == 0.0) {
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .re = 0.0;
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .im = ai / Temp->Space.N;
        } else {
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .re = ar / Temp->Space.N;
          Sol->Psio
              ->data[(static_cast<int32_T>(q) + Sol->Psio->size[0] * i) - 1]
              .im = ai / Temp->Space.N;
        }
      }
      ar = 2.0 * Temp->Space.N;
      if (Temp->Space.N + 1.0 > ar) {
        inembed = 0;
        b_i2 = 0;
      } else {
        inembed = static_cast<int32_T>(Temp->Space.N + 1.0) - 1;
        b_i2 = static_cast<int32_T>(ar);
      }
      for (i = 0; i < b_i2 - inembed; i++) {
        if (validLaunchParams) {
          hipMemcpy(F_e, *gpu_F_e, 8192ULL, hipMemcpyDeviceToHost);
        }
        ar = F_e[inembed + i].re;
        validLaunchParams = false;
        ai = F_e[inembed + i].im;
        if (ai == 0.0) {
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .re = ar / Temp->Space.N;
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .im = 0.0;
        } else if (ar == 0.0) {
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .re = 0.0;
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .im = ai / Temp->Space.N;
        } else {
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .re = ar / Temp->Space.N;
          Sol->Psie
              ->data[(static_cast<int32_T>(q) + Sol->Psie->size[0] * i) - 1]
              .im = ai / Temp->Space.N;
        }
      }
      Sol->t->data[static_cast<int32_T>(q) - 1].re =
          Temp->Par.dt * (static_cast<real_T>(ni) + 1.0);
      Sol->t->data[static_cast<int32_T>(q) - 1].im = 0.0;
      //             if 10*log10(sum(abs(Sol.Psio(ind_s,2:end)).^2)) <= -100 &&
      //             (mod(ind_s,2) == 0)
      //                    Sol.Psio(ind_s+1:end,:) = [];
      //                   Sol.Psie(ind_s+1:end,:) = [];
      //                  Sol.t(ind_s+1:end)     = [];
      //                 break;
      //        end
    }
  }
  emxFree_creal_T(&b_d);
  emxFree_cuint8_T(&d);
  emlrtHeapReferenceStackLeaveFcnR2012b(emlrtRootTLSGlobal);
  gpuEmxFree_cuint8_T(&gpu_d);
  hipFree(gpu_Temp);
  hipFree(*gpu_exp_plus_omega_contents);
  hipFree(*gpu_exp_minus_omega_contents);
  hipFree(*gpu_F_e);
  hipFree(*gpu_y);
  hipFree(gpu_i);
  gpuEmxFree_creal_T(&b_gpu_d);
  hipFree(*gpu_E_temp);
  hipFree(*gpu_Psio);
  hipFree(*gpu_Psie);
  hipFree(*gpu_k_e);
  hipFree(*gpu_x);
  hipFree(*b_gpu_x);
  hipFree(*b_gpu_y);
  hipFree(*b_gpu_E_temp);
  hipFree(gpu_q);
}

// End of code generation (Chi23_Runge_Kuarong_GPU.cu)

//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// Chi23_Runge_Kuarong_GPU_terminate.cu
//
// Code generation for function 'Chi23_Runge_Kuarong_GPU_terminate'
//

// Include files
#include "Chi23_Runge_Kuarong_GPU_terminate.h"
#include "Chi23_Runge_Kuarong_GPU_data.h"
#include "_coder_Chi23_Runge_Kuarong_GPU_mex.h"
#include "rt_nonfinite.h"

// Function Definitions
void Chi23_Runge_Kuarong_GPU_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void Chi23_Runge_Kuarong_GPU_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(static_cast<uint32_T>(errCode),
                       (char_T *)hipGetErrorName(errCode),
                       (char_T *)hipGetErrorString(errCode),
                       (char_T *)"SafeBuild", emlrtRootTLSGlobal);
  }
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (Chi23_Runge_Kuarong_GPU_terminate.cu)
